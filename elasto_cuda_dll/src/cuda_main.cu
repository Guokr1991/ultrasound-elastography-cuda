#include "hip/hip_runtime.h"
#include  "cuda_main.cuh"


#ifdef  _CUDA_MAIN_CUH
#include  "SysConfig.h"
#include  "CElasto.h"
#include  "FileUtility.h"

#include <fstream>
#include <string>
#include <iostream>
#include <time.h>
#include <>
#include <hip/device_functions.h>
#include <math_functions.h>
#include <string.h>
#include <cstdio>

#endif


//�ں˺�����device����


//��Ҫ���ģ��ÿ�GPUоƬֻ��֧��1024��threads per  block !!!      changed  by  wong   2016/06/08

//�����������˲���    ���˲�           changed  by  wong    2016/5/13

__global__ void Bandpass_front_1(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)
{
	//float muData[40];

	//k  = blockIdx.x   i = blockIdx.y;
	float data_sum;

	float data_1;

	data_sum = 0.0;

	//	__shared__     float     data_sum[8192];



	/*for (int i = 0; i < iParaLen; i++)
	{
	muData[i] = *(param + i);
	}*/

	if (threadIdx.x <= iParaLen - 1)                                         //������ĿС�ڵ��ڳ�ͷ��Ŀ
	{

		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);      //b(0)*x(n-0)+b(1)*x(n-1)+...+b(n)*x(0)   

			data_sum += (data_1*param[i]);

		}

		data_1 = *(tInput + blockIdx.x * iWidth);                          // x(0)


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{
			data_sum += (data_1*param[j]);                                 //b(n+1)*x(0)+...+b(nb-2)*x(0)
		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x) = data_sum;



	}
	else                                                                  //������Ŀ���ڳ�ͷ��Ŀ            
	{
		//data_1 = (tInput + blockIdx.x*iWidth + blockIdx.y - threadIdx.x)->x;
		for (int i = 0; i <= iParaLen - 1; i++)
		{

			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);   //b(0)*x(n-0)+b(1)*x(n-1)+...+b(nb-2)*x(n-(nb-2))  

			data_sum += (data_1*param[i]);

		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x)= data_sum;

	}

}



//�����������˲���   ���˲�                          changed  by  wong     2016/5/13
__global__ void Bandpass_back_1(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)
{
	/*
	if (threadIdx.x < iParaLen - 1)                                     //  n  >= nb-1
	{
	return;                                                            //  �˴�������     changed  by  wong
	}
	//   changed   by  wong     2016/5/11
	*/                                                                 // �˴�û�п���С��nb-1���������Ҫ�������


	float data_1;

	float data_sum;

	data_sum = 0.0;



	if (threadIdx.x <= iParaLen - 1)   {                                //  ���ݳ���С�ڵ����˲�����ͷ����


		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);      //

			data_sum += (data_1*param[i]);

		}


		data_1 = *(tInput + blockIdx.x * iWidth + iWidth - 1);                        //  x(N-1) 


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{

			data_sum += (data_1*param[j]);                                            // b(n+1)*x(N-1)+...+b(nb-1)*x(N-1) 

		}


		*(tOutPut + blockIdx.x * iWidth + threadIdx.x) = data_sum;                 // y(n)

	}

	else    {                                                                       //���ݳ��ȴ����˲�����ͷ����         


		for (int i = 0; i <= iParaLen - 1; i++)
		{
			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);

			data_sum += (data_1*param[i]);                                         //  y(N-1-n) = b(0)*x(N-1-n+0) +b(1)*x(N-1-n+1)+...+b(nb-1)*x(N-1-n+nb-1)

		}

		*(tOutPut + blockIdx.x * iWidth + iWidth - 1 - threadIdx.x)   = data_sum;




	}


}


//changed   by  wong      2016/5/13

__device__      void   xcorr_cuda(const  Complex* templateMat_startID, const Complex* objectMat_startID, Complex*resultMat_startID)     {


	for (int i = 0; i < 101; i++)   {


		Complex     sum_object = 0;

		Complex     frac_object = 0;


		Complex     pow_template = 0;


		Complex     pow_object = 0;


		Complex     result = 0;


		//sum_object 

		for (int j = 0; j < 100; j++)  {


			sum_object += *(objectMat_startID + i + j);


		}

		//  ave

		Complex   ave_object = sum_object / 200;


		//fraction

		for (int j = 0; j < 100; j++)  {

			Complex    tmp = *(templateMat_startID + j) *  (*(objectMat_startID + i + j) - ave_object);


			frac_object += tmp;

		}


		//pow   temp

		for (int j = 0; j < 100; j++)  {


			pow_template += *(templateMat_startID + j) * *(templateMat_startID + j);

		}

		//pow   objectMat 

		for (int j = 0; j < 100; j++)  {


			pow_object += *(objectMat_startID + i + j)* * (objectMat_startID + i + j);

		}

		//result

		result = sqrt(pow_template*pow_object);

		//output

		*(resultMat_startID + i) = frac_object / result;

	}


}


//changed   by  wong    2016/5/13

__device__      void   minMax_cuda(Complex*resultMat_startID, Complex* min_value, Complex*  max_value, int  max_location)   {

	int      max_loc_temp = 0;

	int      min_loc_temp = 0;

	Complex* max_temp     = 0;

	Complex* min_temp    = 0;

	//�����ֵ��λ��

	for (int i = 0; i < 101; i++)  {

		if (*(resultMat_startID + i) >= *max_temp)  {

			*max_temp = *(resultMat_startID + i);

			max_loc_temp = i;

		}

	}

	//����Сֵ��λ��

	for (int i = 0; i < 101; i++)  {

		if (*(resultMat_startID + i) <= *min_temp)  {

			*min_temp = *(resultMat_startID + i);

			min_loc_temp = i;
		}


	}

	//���

	*min_value = *min_temp;

	*max_value = *max_temp;

	max_location = max_loc_temp;

}


 
//changed  by   wong      2016/5/17

__device__    void    interp_cuda(Complex*resultMat_startID, int  max_loc, Complex*max_value, int multiWin, int  winSize, Complex*  displace)     {

	Complex*pre  = resultMat_startID + max_loc - 1;

	Complex*next = resultMat_startID + max_loc + 1;


	*displace   = (multiWin - 1) * winSize / 2 - max_loc - (*pre - *next) / (2 * (*pre - 2 * *max_value + *next));


}





//���Ϊλ��299*799����

__global__   void  displacement_api_cuda(Complex*disInputCuda, int rows, int cols, int  multiWin, int winSize, int  stepSize, templateMat*templateMatShare, objectMat* objectMatShare, resultMat*resultMatShare, Complex*min, Complex*max, int*max_location, Complex* displacement )      {


	int   out_offset = blockIdx.x *blockDim.x + threadIdx.x;                     // ���λ�ƾ���ƫ��ֵ

	int    bid       = blockIdx.x ;                                              //  ��Ӧblock ID 
	
	int    tid       = threadIdx.x;                                             //   ��Ӧthread ID 


   //����ʹ��3D���飬��Ϊ�����ڴ治���ã�ֻ��49152���ֽ�each block 

	//�����ڴ�

	//����ȫ���ڴ棡����

//	__shared__     Complex*     templateMatShare[THREAD_NUM];        //100�׵�ַ

//	__shared__     Complex*     objectMatShare[THREAD_NUM];          //200�׵�ַ

//	__shared__     Complex*     resultMatShare[THREAD_NUM];          //101�׵�ַ 


//	__shared__     templateMat   templateMatShare[THREAD_NUM];

//	__shared__     objectMat     objectMatShare[THREAD_NUM];

//	__shared__     resultMat     resultMatShare[THREAD_NUM];


//	  Complex*templateMatShare;                          //   ģ���ڴ���GPU����         ���Ǿֲ�����                  


//	  Complex*objectMatShare;                           //    Ŀ���ڴ���GPU����         ���Ǿֲ�����


//	  Complex*resultMatShare;                           //    ƥ������GPU����         ���Ǿֲ�����




//	hipMalloc(&templateMatShare, winSize* sizeof(Complex));             //ģ�����


//	hipMalloc(&objectMatShare, winSize*multiWin* sizeof(Complex));     //Ŀ�����


//	hipMalloc(&resultMatShare, (winSize + 1)* sizeof(Complex));        //�������




//	 templateMatShare[out_offset].elem   = (Complex*)(disInputCuda + blockIdx.x*cols + (multiWin - 1) * winSize / 2 + threadIdx.x * stepSize);
		







	    Complex*templateMatID;                               //ID


	  Complex*objectMatID;                                //ID



	//12784�ֽ�

//	__shared__     Complex*     min[THREAD_NUM];

//	__shared__     Complex*     max[THREAD_NUM];

//	__shared__      int        max_location[THREAD_NUM];

//	__shared__    Complex*     displacement[THREAD_NUM];




	//׼���������      �߳̿������� �����ù����ڴ�


	//	(templateMat*)(templateMat_startID + threadIdx.x)->elem = (Complex*)(disInputCuda + blockIdx.x*cols + (multiWin - 1) * winSize / 2 + threadIdx.x * stepSize);

	      templateMatID = (Complex*)(disInputCuda + blockIdx.x*cols + (multiWin - 1) * winSize / 2 + threadIdx.x * stepSize);


//		  hipMemcpy(templateMatShare[out_offset].elem, templateMatID, winSize, hipMemcpyDeviceToDevice);

		  for (int i = 0; i < 100; i++) {

			  templateMatShare[out_offset].elem[i] = *(templateMatID+i);

		  }

 // templateMatShare[threadIdx.x].elem = (Complex*)(disInputCuda + blockIdx.x*cols + (multiWin - 1) * winSize / 2 + threadIdx.x * stepSize);


		  objectMatID   = (Complex*)(disInputCuda + (blockIdx.x + 1)*cols + threadIdx.x * stepSize);


//		  hipMemcpy(objectMatShare[out_offset].elem, objectMatID, winSize*multiWin, hipMemcpyDeviceToDevice);

             
		  for (int i = 0; i < 200; i++)  {
		  
			  objectMatShare[out_offset].elem[i] = *(objectMatID+i);
		  
		  }
  


	//		objectMat_startID[threadIdx.x].elem   = disInputCuda + (blockIdx.x + 1)*cols + threadIdx.x * stepSize;



//	__syncthreads();

//	hipDeviceSynchronize();

//	hipDeviceSynchronize();



	//�������

		  xcorr_cuda(templateMatShare[out_offset].elem, objectMatShare[out_offset].elem, resultMatShare[out_offset].elem);


//		__syncthreads();
//	hipDeviceSynchronize();


	//�������ֵ

		  minMax_cuda(resultMatShare[out_offset].elem, &min[out_offset], &max[out_offset], max_location[out_offset]);


//	__syncthreads();

//	hipDeviceSynchronize();

	//��ֵ

		  interp_cuda(resultMatShare[out_offset].elem, max_location[out_offset], &max[out_offset], multiWin, winSize, &displacement[out_offset]);


//	__syncthreads();

//	hipDeviceSynchronize();


	//ȥ����


	//λ�Ƶ���


	//��ֵ�˲�


	//�����ֵ

	//  *��disOutputCuda+bid��     =    displacement   ��

//	disOutputCuda[out_offset] = *displacement[threadIdx.x];


}



//ȥ����        changed  by wong    2016/5/18

__global__  void   remove_singular_cuda(Complex*disOutputCuda, Complex*singularOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                         // ���λ�ƾ���ƫ��ֵ

	int   offrow = (blockIdx.x - 1)*blockDim.x + threadIdx.x;                   // ��һ�����λ�ƾ���ƫ��ֵ

	int    bid   = blockIdx.x;                                                  //  block   id

	int    tid   = threadIdx.x;                                                //  thread  id             


	if (bid > 0 && (disOutputCuda[offset] > 12))  {

		singularOutputCuda[offset] = disOutputCuda[offrow];

	}

	else  {

		singularOutputCuda[offset] = disOutputCuda[offset];

	}


}


//λ�Ƶ���       changed   by  wong    2016/5/18

__global__   void   displace_add_cuda(Complex*singularOutputCuda, Complex*addOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                        // ���λ�ƾ���ƫ��ֵ

	int   offrow = (blockIdx.x - 1)*blockDim.x + threadIdx.x;                   // ��һ�����λ�ƾ���ƫ��ֵ

	int    bid   = blockIdx.x;                                                  // block   id

	int    tid   = threadIdx.x;                                                 // thread  id             

	if (bid > 0)  {

		addOutputCuda[offset] = singularOutputCuda[offset] + singularOutputCuda[offrow];


	}

	else   {

		addOutputCuda[offset] = singularOutputCuda[offset];

	}



}



//������չN-1�У���������

__global__   void   extend_data_cuda(Complex*addOutputCuda, Complex*extendOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                        // ���λ�ƾ���ƫ��ֵ

	int    bid   = blockIdx.x;                                                  // block   id

	int    tid   = threadIdx.x;                                                 // thread  id   


	if (tid<N - 1)  {

		extendOutputCuda[offset] = 0;                                          //  extend  0

	}

	else
	{

		int   extoff = blockIdx.x *(blockDim.x - (N - 1)) + threadIdx.x - (N - 1);

		extendOutputCuda[offset] = addOutputCuda[extoff];

	}

}


//���ۼ�ƽ��   

__global__ void  smooth_filter_cuda(Complex*extendOutputCuda, Complex* smoothOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                        // ���λ�ƾ���ƫ��ֵ

	int   extbase = blockIdx.x*(blockDim.x + N - 1) + threadIdx.x;              // ��ַ

	int    bid = blockIdx.x;                                                    // block   id

	int    tid = threadIdx.x;                                                   // thread  id  


	Complex*  sum = 0;


	for (int i = extbase; i < extbase + N; i++)  {


		*sum += extendOutputCuda[i];


	}

	smoothOutputCuda[offset] = *sum / N;

}




__global__  void   timeField_filter_cuda(const Complex* smoothOutputCuda, const float* param,  const int  steps, Complex* timeFilterOutputCuda)    {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                        // ���λ�ƾ���ƫ��ֵ


	int    bid = blockIdx.x;                                                    // block   id

	int    tid = threadIdx.x;                                                   // thread  id     

	Complex  sum_temp = 0;

	float    coeff   = 0;

	for (int i = 0; i <= bid; i++)   {

		if ((bid - i) < steps)

			coeff = param[bid - i];

		else

			coeff = param[0];


		sum_temp += smoothOutputCuda[i*blockDim.x + threadIdx.x] * coeff;


	}

	timeFilterOutputCuda[offset] = sum_temp;

}






bool    CudaMain::isAvailable()  {

	int   count = 0;

	printf("Start to detecte devices.........\n");                   //  ��ʾ��⵽���豸��

	hipGetDeviceCount(&count);                                     //   �������������ڵ���1.0���豸��

	if (count == 0){

		fprintf(stderr, "There is no device.\n");

		return false;

	}


	printf("%d device/s detected.\n", count);                      //   ��ʾ��⵽���豸��


	int i;

	for (i = 0; i < count; i++){                                  //  ������֤��⵽���豸�Ƿ�֧��CUDA

		hipDeviceProp_t prop;

		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {  //  ����豸���Բ���֤�Ƿ���ȷ

			if (prop.major >= 1)                                 //  ��֤�����������������������ĵ�һλ���Ƿ����1

			{
				printf("Device %d: %s supports CUDA %d.%d.\n", i + 1, prop.name, prop.major, prop.minor);//��ʾ��⵽���豸֧�ֵ�CUDA�汾
				break;


			}
		}
	}

	if (i == count) {                                         //   û��֧��CUDA1.x���豸
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);                                       //    �����豸Ϊ�����̵߳ĵ�ǰ�豸

	return true;

}





CudaMain::CudaMain()  {

	


	cpu_inputMat      = NULL;
	
	cpu_SplineOutMat  = NULL ;

	cpu_RadonMat      = NULL;

	cpu_WaveRate      = 0  ;

	mallocFlag        = false;

	cpu_config        = new   ConfigParam ;

	cpu_disMat        = NULL;



//	memset(cpu_config, 0, sizeof(cpu_config));




	inputMat         = NULL;

	zeroFilterMat    = NULL;

	frontFilterMat   = NULL;

	disOutput        = NULL;

	bandfilterParam  = NULL;

	lowfilterParam   = NULL;

	matchfilterParam = NULL;

	lowFrontMat      = NULL;

	lowBackMat       = NULL;

	singularOutputCuda = NULL;

	addOutputCuda      = NULL;

	extendOutputCuda   = NULL;


	radonIn          = NULL;

	radonOut         = NULL;


}




CudaMain :: ~CudaMain()  {

	freeMem(); 



}



void   CudaMain::inputConfigParam( ConfigParam*config) {



	cpu_config = config;


}



void  CudaMain::inputRfData(const EInput& in) {     //�������ݵ�cpu_inputMat��

	float* input = in.pDatas;


	for (int i = 0; i < cpu_inputMat->rows; i++)
	{
		for (int j = 0; j < cpu_inputMat->cols; j++)
		{
			*(static_cast<float*>(static_cast<void*>(CV_MAT_ELEM_PTR(*cpu_inputMat, i, j)))) = input[i * cpu_inputMat->cols + j];
		}
	}


}




void  CudaMain::getbandFilterParam(std::string paramFileName) {

	if (paramFileName.size() == 0)
	{
		exit(1);
	}

	std::fstream paramFile(paramFileName.c_str());

	if (!paramFile)
	{
		exit(1);
	}

	float tmp;

	std::string str;


	cpu_bandfilterParam.clear();

	while (!paramFile.eof())
	{
		paramFile >> tmp;
		cpu_bandfilterParam.push_back(tmp);
	}
	paramFile.close();



}


void   CudaMain::getlowFilterParam(std::string paramFileName) {

	if (paramFileName.size() == 0)
	{
		exit(1);
	}

	std::fstream paramFile(paramFileName.c_str());

	if (!paramFile)
	{
		exit(1);
	}

	float tmp;

	std::string str;


	cpu_lowfilterParam.clear();

	while (!paramFile.eof())
	{
		paramFile >> tmp;
		cpu_lowfilterParam.push_back(tmp);
	}
	paramFile.close();


}



void  CudaMain::getmatchFilterParam(std::string paramFileName) {

	if (paramFileName.size() == 0)
	{
		exit(1);
	}

	std::fstream paramFile(paramFileName.c_str());

	if (!paramFile)
	{
		exit(1);
	}

	float tmp;

	std::string str;


	cpu_matchfilterParam.clear();

	while (!paramFile.eof())
	{
		paramFile >> tmp;
		cpu_matchfilterParam.push_back(tmp);
	}
	paramFile.close();

}











void  CudaMain::mallocMem(void)  {

	mallocMats();

	mallocGPUMem();

	

}



void CudaMain::freeMem(void)  {

	freeMats();
   
	deleteGPUMem();
}





void   CudaMain::mallocGPUMem() {



	int  MatRows = cpu_config->shearFrameLineNum;

	int  MatCols = cpu_config->sampleNumPerLine ;

	int windowHW = cpu_config->windowHW;

	int maxLag   = cpu_config->maxLag;

	int step     = cpu_config->step;


	int interpnum  = cpu_config->fitline_pts;

	int iBPParaLen = 40;                                                      // bandpassfilter�ĳ��ȣ�

	iBPParaLen     = (iBPParaLen > cpu_bandfilterParam.size()) ? iBPParaLen : cpu_bandfilterParam.size();


	int iLPParaLen = 40;                                                      // lowpassfilter�ĳ��ȣ�

	iLPParaLen     = (iBPParaLen > cpu_lowfilterParam.size()) ? iBPParaLen : cpu_lowfilterParam.size();


	int iMHParaLen = 40;                                                      // matchfilter�ĳ��ȣ�

	iMHParaLen    = (iBPParaLen > cpu_matchfilterParam.size()) ? iBPParaLen : cpu_matchfilterParam.size();






	if (MatRows == 0 || MatCols == 0)
	{

		printf("  row  and col  is zero! call InputConfigParas first!\n");
		return;

	}

	hipError_t cudaStatus = hipSetDevice(0);                                 // 0��titan�Կ�

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}



	int  multiWin = 2;                                                      //  �󴰿ڶ�С���ڵı���

	int cxorrLines = MatRows - 1;                                           //  λ�ƾ������ɨ������Ŀ        299

	int iOutRows = (MatCols - multiWin*windowHW) / step;                    //  λ�ƾ��������Ҫƥ��Ķ���     799 

	int extRows = iOutRows + N - 1;                                         //  ��չ����  799+100-1

	hipMalloc(&disOutput, cxorrLines *iOutRows* sizeof(Complex));          //  λ�ƾ���GPU�ڴ����


//	hipMalloc(&templateMatShare, cxorrLines *iOutRows* sizeof(templateMat));       //ģ�������GPUȫ���ڴ����


//	hipMalloc(&objectMatShare, cxorrLines *iOutRows* sizeof(objectMat));         //Ŀ�������GPUȫ���ڴ����


//	hipMalloc(&resultMatShare, cxorrLines *iOutRows* sizeof(resultMat));        //���������GPUȫ���ڴ����






	hipMalloc(&singularOutputCuda, cxorrLines *iOutRows* sizeof(Complex)); // ȥ���������GPU�ڴ����


	hipMalloc(&addOutputCuda, cxorrLines *iOutRows* sizeof(Complex));      // λ�Ƶ�����GPU�ڴ����


	hipMalloc(&extendOutputCuda, cxorrLines *extRows* sizeof(Complex));     // ��չ������GPU�ڴ����



	hipMalloc(&inputMat, MatRows * MatCols * sizeof(Complex));             //   ���������GPU�϶�Ӧ���ڴ棻


	hipMalloc(&zeroFilterMat, MatRows * MatCols * sizeof(Complex));       //   ��ͨ����λ�˲������GPU�ڴ���䣻


	hipMalloc(&frontFilterMat, MatRows * MatCols * sizeof(Complex));     //  ��ͨ����λ���˲���GPU�ڴ����



	hipMalloc(&lowBackMat, cxorrLines * iOutRows * sizeof(Complex));       //   ��ͨ����λ�˲������GPU�ڴ���䣻


	hipMalloc(&lowFrontMat, cxorrLines * iOutRows * sizeof(Complex));     //  ��ͨ����λ���˲���GPU�ڴ����

	


	hipMalloc(&bandfilterParam, iBPParaLen * sizeof(float));                // iBPParaLen�˲�������40


	hipMalloc(&lowfilterParam, iLPParaLen * sizeof(float));                // iLPParaLen�˲�������40


	hipMalloc(&matchfilterParam, iMHParaLen * sizeof(float));              // iMHParaLen�˲�������40



	int RadonInputCols      = 1961;                                     // 1961

	int RadonInputRows      = 4;                                       // 4

	hipMalloc(&radonIn, sizeof(float) * RadonInputCols * RadonInputRows);                //�����任GPU����

	hipMalloc(&radonOut, sizeof(float) * RadonInputCols * (RadonInputCols - 1));        //�����任GPU���  


	mallocFlag             = true;




}





void  CudaMain::deleteGPUMem()  {


	if (inputMat != NULL)
	{
		hipFree(inputMat);

		inputMat = NULL;
	}

	
	if (zeroFilterMat != NULL)
	{
		hipFree(zeroFilterMat);
		zeroFilterMat = NULL;
	}


	if (frontFilterMat != NULL)
	{
		hipFree(frontFilterMat);
		frontFilterMat = NULL;
	}


	if (lowBackMat != NULL)
	{
		hipFree(lowBackMat);
		lowBackMat = NULL;
	}


	if (lowFrontMat != NULL)
	{
		hipFree(lowFrontMat);
		lowFrontMat = NULL;
	}




	if (disOutput != NULL)
	{
		hipFree(disOutput);
		disOutput = NULL;
	}




	if (singularOutputCuda != NULL)
	{
		hipFree(singularOutputCuda);

		singularOutputCuda = NULL;
	}


	if (addOutputCuda != NULL)
	{
		hipFree(addOutputCuda);

		addOutputCuda = NULL;
	}


	if (extendOutputCuda != NULL)
	{
		hipFree(extendOutputCuda);

		extendOutputCuda = NULL;
	}







	if (bandfilterParam != NULL)
	{
		hipFree(bandfilterParam);
		bandfilterParam = NULL;
	}

	
	if (lowfilterParam != NULL)
	{
		hipFree(lowfilterParam);
		lowfilterParam = NULL;
	}



	if (matchfilterParam != NULL)
	{
		hipFree(matchfilterParam);
		matchfilterParam = NULL;
	}






	if (radonIn != NULL)
	{
		hipFree(radonIn);
	}

	if (radonIn != NULL)
	{
		hipFree(radonIn);
	}


	hipDeviceReset();

	mallocFlag = false;


}




void  CudaMain::mallocMats() {


	cpu_inputMat    =   cvCreateMat(cpu_config->shearFrameLineNum, cpu_config->sampleNumPerLine, CV_32FC1);         //�������

	int  MatRows    = cpu_config->shearFrameLineNum;

	int  MatCols    = cpu_config->sampleNumPerLine;

	int windowHW    = cpu_config->windowHW;

	int maxLag      = cpu_config->maxLag;

	int step        = cpu_config->step;


	int  multiWin   = 2;                                                    //  �󴰿ڶ�С���ڵı���

	int cxorrLines  = MatRows - 1;                                         //   λ�ƾ������ɨ������Ŀ        299

	int iOutRows    = (MatCols - multiWin*windowHW) / step;               //    λ�ƾ��������Ҫƥ��Ķ���     799 

	cpu_disMat      = cvCreateMat(cxorrLines, iOutRows, CV_32FC1);       //     λ�ƾ���     


	cpu_SplineOutMat = cvCreateMat(1962, 4, CV_32FC1);                  //    SplineOutMat��������ڻ�ͼ���ȽϽ��  

		
	cpu_RadonMat    = cvCreateMat(1962, 4, CV_32FC1);                  //     radon������Ƚϼ�����  



	mallocFlag     = false; 
	

//	cpu_config     = (ConfigParam*)malloc(1 * sizeof(ConfigParam));     

	
//	memset(cpu_config, 0, sizeof(cpu_config));

}



void   CudaMain::freeMats() {

	if (cpu_inputMat != NULL)
	{
		cvReleaseMat(&cpu_inputMat);
		cpu_inputMat = NULL;
	}
	

	if (cpu_disMat != NULL)
	{
		cvReleaseMat(&cpu_disMat);
		cpu_disMat = NULL;
	}
	

	if (cpu_SplineOutMat != NULL)
	{
		cvReleaseMat(&cpu_SplineOutMat);
		cpu_SplineOutMat = NULL;
	}

	if (cpu_RadonMat != NULL)
	{
		cvReleaseMat(&cpu_RadonMat);
		cpu_RadonMat = NULL;
	}
	

	mallocFlag = NULL;


	free(cpu_config);


	cpu_config = NULL;

	  
}










CvMat*  CudaMain::bandpassFilt_cuda(CvMat* rawMat)  {


	Complex* h_MatData = (Complex*)rawMat->data.fl;

	hipMemsetAsync(frontFilterMat, 0, sizeof(Complex)*rawMat->cols*rawMat->rows);

	hipMemcpyAsync(zeroFilterMat, h_MatData, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyHostToDevice);    //����CPU��RF���ݵ�GPU

	int steps = cpu_bandfilterParam.size();

	hipMemcpyAsync(bandfilterParam, &cpu_bandfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);                  //����CPU�г�ͷ���ݵ�GPU 





	dim3 blockID, threadID;

	blockID.x  = rawMat->rows;

	threadID.x = rawMat->cols;

	hipDeviceSynchronize();

	Bandpass_front_1 <<<blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);

	hipDeviceSynchronize();


	hipMemcpy(zeroFilterMat, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToDevice);


	Bandpass_back_1 << <blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);


	hipDeviceSynchronize();

	   
	hipFree(bandfilterParam);

	hipMemcpy(h_MatData, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

	hipFree(zeroFilterMat);

	hipFree(frontFilterMat);


	SaveDataFile("bpfilt.dat", rawMat);


	return rawMat;


}







void  CudaMain::zeroFilter_cuda(CvMat* rawMat, Complex*filterOutput) {







}





CvMat*  CudaMain::computeDisplacement_cuda(CvMat* filtOutMat, int  multiWin, int winSize, int stepSize){

//	CvMat*outputMat = 0;

	int     WinNum    = (filtOutMat->cols - multiWin*winSize) / stepSize;       //  һάλ�ƾ���

	Complex* hInput   = (Complex*)filtOutMat->data.fl;                         //   ����λ�ã�

	//Complex*hOutput = (Complex*)outputMat->data.fl;                        //   ����λ�ã�


	hipMemcpy(inputMat, hInput, filtOutMat->cols*filtOutMat->rows*sizeof(Complex), hipMemcpyHostToDevice);   //  CPU-GPU

	dim3 dBlock;

	dim3 dThread;

	dBlock.x = filtOutMat->rows - 1;                                 // ����������� ,����        299

	dThread.x = WinNum;                                             // ����������� , �߳���      799


//	__device__   Complex*templateMatShare;                          //   ģ���ڴ���GPU����         ���Ǿֲ�����                  


//	__device__   Complex*objectMatShare;                           //    Ŀ���ڴ���GPU����         ���Ǿֲ�����


//	__device__   Complex*resultMatShare;                           //    ƥ������GPU����         ���Ǿֲ�����




	templateMat*templateMatShare;                                 //   ģ���ڴ���GPU���� 


	objectMat* objectMatShare;                                   //    Ŀ���ڴ���GPU���� 



	resultMat*resultMatShare;                                   //    ƥ������GPU���� 



	Complex*      min;


	Complex*      max;

	int*          max_location;


	Complex*      displacement;








	hipMalloc(&templateMatShare, dBlock.x*dThread.x* sizeof(templateMat));             //ģ�������GPUȫ���ڴ����


	hipMalloc(&objectMatShare,  dBlock.x*dThread.x* sizeof(objectMat));               //Ŀ�������GPUȫ���ڴ����


	hipMalloc(&resultMatShare,  dBlock.x*dThread.x* sizeof(resultMat));             //���������GPUȫ���ڴ����



	hipMalloc(&min, dBlock.x*dThread.x* sizeof(Complex));                           // min��GPUȫ���ڴ����


	hipMalloc(&max, dBlock.x*dThread.x* sizeof(Complex));                          // max��GPUȫ���ڴ����


	hipMalloc(&max_location, dBlock.x*dThread.x* sizeof(int));                     // max_location��GPUȫ���ڴ����


	hipMalloc(&displacement, dBlock.x*dThread.x* sizeof(Complex));                // max_location��GPUȫ���ڴ����



	
	//��λ�ƾ���  

	displacement_api_cuda << < dBlock, dThread >> >   (inputMat, filtOutMat->rows, filtOutMat->cols, multiWin, winSize, stepSize, templateMatShare, objectMatShare, resultMatShare, min, max, max_location, displacement);

	hipDeviceSynchronize();


	hipFree(templateMatShare);

	hipFree(objectMatShare);

	hipFree(resultMatShare);

	hipFree(min);

	hipFree(max);

	hipFree(max_location);








	//ȥ����                                   

	remove_singular_cuda << <dBlock, dThread >> >   (displacement, singularOutputCuda);

	hipDeviceSynchronize();

	//λ�Ƶ���                 

	displace_add_cuda << <dBlock, dThread >> >  (singularOutputCuda, addOutputCuda);

	hipDeviceSynchronize();

	//ǰN-1�в�0    

	int  ext_threads = dThread.x + N - 1;

	extend_data_cuda << < dBlock, ext_threads >> > (addOutputCuda, extendOutputCuda);

	hipDeviceSynchronize();

	hipFree(addOutputCuda);

	//ƽ���˲�  

	smooth_filter_cuda << <dBlock, dThread >> >   (extendOutputCuda, disOutput);

	hipDeviceSynchronize();

	hipFree(extendOutputCuda);

	//ʱ���˲���ƥ���˲���50Hz��ǿ     ���� param, iParaLen, steps ʹ�ó����ڴ� 

	int steps = cpu_matchfilterParam.size();

	hipMemcpyAsync(matchfilterParam, &cpu_matchfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);             //����CPU�г�ͷ���ݵ�GPU 


	timeField_filter_cuda << <dBlock, dThread >> > (disOutput, matchfilterParam,  steps, singularOutputCuda);

	hipDeviceSynchronize();

	hipFree(disOutput);

	//��GPU������CPU�ڴ�


	hipMemcpy(hInput, singularOutputCuda, dBlock.x  * dThread.x*sizeof(Complex), hipMemcpyDeviceToHost);   //  GPU-CPU


	hipFree(singularOutputCuda);

	return    filtOutMat;



}






void   CudaMain::zeroDisplacement_cuda(CvMat* inputMat, int  multiWin, int winSize, int stepSize, Complex*disOutput){





}





CvMat*  CudaMain::lowpassFilt_cuda(CvMat* disMat)  {


	Complex* h_MatData = (Complex*)disMat->data.fl;

	hipMemsetAsync(lowBackMat, 0, sizeof(Complex)*disMat->cols*disMat->rows);

	hipMemcpyAsync(lowFrontMat, h_MatData, sizeof(Complex)*disMat->cols*disMat->rows, hipMemcpyHostToDevice);           //����CPU��RF���ݵ�GPU

	int steps = cpu_lowfilterParam.size();

	hipMemcpyAsync(lowfilterParam, &cpu_lowfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);                  //����CPU�г�ͷ���ݵ�GPU 





	dim3 blockID, threadID;

	blockID.x = disMat->rows;

	threadID.x = disMat->cols;

	hipDeviceSynchronize();

	Bandpass_front_1 << <blockID, threadID >> >(lowFrontMat, disMat->cols, lowfilterParam, steps, lowBackMat);

	hipDeviceSynchronize();


	hipMemcpy(lowFrontMat, lowBackMat, sizeof(Complex)*disMat->cols*disMat->rows, hipMemcpyDeviceToDevice);


	Bandpass_back_1 << <blockID, threadID >> >(lowFrontMat, disMat->cols, lowfilterParam, steps, lowBackMat);


	hipDeviceSynchronize();


	hipFree(lowfilterParam);

	hipMemcpy(h_MatData, lowBackMat, sizeof(Complex)*disMat->cols*disMat->rows, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

	hipFree(lowFrontMat);

	hipFree(lowBackMat);


	return disMat;


}









void  CudaMain::process(const EInput &input, EOutput& output) {

//	mallocMem();                                                                                 // �����ڴ�

//	inputRfData(input);                                                                          // ��ȡRF����,��cpu_inputMat 

//	inputConfigParam(config);                                                                    // ���ò��� ��cpu_config

// getFilterParam(config->bpfilt_file);                                                          // ��ȡ�˲�����ͨ��������cpu_filterParam




	   bandpassFilt_cuda(cpu_inputMat);                                                          //    ��ͨ�˲�       


	   int  multiWin    = 2;

	   int winSize      = cpu_config->windowHW;

	   int  stepSize    = cpu_config->step;


	   
	   cpu_disMat       = computeDisplacement_cuda(cpu_inputMat,  multiWin,  winSize, stepSize);   //  λ�Ƽ���       
	

	   lowpassFilt_cuda(cpu_disMat);                                                              //    ��ͨ�˲�     




	   










}



