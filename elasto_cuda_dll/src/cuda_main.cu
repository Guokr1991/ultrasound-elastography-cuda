#include "hip/hip_runtime.h"
#include  "cuda_main.cuh"


#ifdef  _CUDA_MAIN_CUH
#include  "SysConfig.h"
#include  "CElasto.h"
#include  "FileUtility.h"
#include <fstream>
#include <string>
#include <iostream>
#include <time.h>
#include <>
#include <hip/device_functions.h>
#include <math_functions.h>
#include  <math.h>
#include <string.h>
#include <cstdio>
#include "opencv/highgui.h"
#include "opencv/cv.h"
#include "ImageFunc.h"


#endif



__global__ void Bandpass_front_1(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)
{
	
	float data_sum;

	float data_1;

	data_sum = 0.0;

	

	if (threadIdx.x <= iParaLen - 1)                                     
	{

		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);      //b(0)*x(n-0)+b(1)*x(n-1)+...+b(n)*x(0)   

			data_sum += (data_1*param[i]);

		}

		data_1 = *(tInput + blockIdx.x * iWidth);                          // x(0)


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{
			data_sum += (data_1*param[j]);                                 //b(n+1)*x(0)+...+b(nb-2)*x(0)
		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x) = data_sum;



	}
	else                                                                        
	{
		//data_1 = (tInput + blockIdx.x*iWidth + blockIdx.y - threadIdx.x)->x;
		for (int i = 0; i <= iParaLen - 1; i++)
		{

			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);   //b(0)*x(n-0)+b(1)*x(n-1)+...+b(nb-2)*x(n-(nb-2))  

			data_sum += (data_1*param[i]);

		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x)= data_sum;

	}

}






__global__ void Bandpass_front_1024(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)     {

	 float data_sum;

	 float data_1;

	 data_sum = 0.0;

	 int   line_serial;

	 int    bid = blockIdx.x;


	   line_serial = bid /16;



	 int  line_mod = bid % 16;






	 if ((0 == line_mod))    {                                                                                    

		 if ((threadIdx.x <= iParaLen - 1))                                                       
		 {

			 for (int i = 0; i <= threadIdx.x; i++)
			 {


				 data_1 = *(tInput + line_serial * iWidth + threadIdx.x - i);                  //b(0)*x(n-0)+b(1)*x(n-1)+...+b(n)*x(0)   

				 data_sum += (data_1*param[i]);

			 }


			 data_1 = *(tInput + line_serial * iWidth);                                      // x(0)


			 for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
			 {
				 data_sum += (data_1*param[j]);                                                    //b(n+1)*x(0)+...+b(nb-2)*x(0)
			 }

			 *(tOutPut + line_serial * iWidth + threadIdx.x) = data_sum;



		 }

		 else  if ((threadIdx.x > iParaLen - 1))   {                                               

			 
			 for (int i = 0; i <= iParaLen - 1; i++)
			 {

				 data_1 = *(tInput + line_serial *iWidth + threadIdx.x - i);                 //b(0)*x(n-0)+b(1)*x(n-1)+...+b(nb-2)*x(n-(nb-2))  

				 data_sum += (data_1*param[i]);

			 }

			 *(tOutPut + line_serial * iWidth + threadIdx.x) = data_sum;


		 }


	 }

	else                                                                                     
	{

		for (int i = 0; i <= iParaLen - 1; i++)
		{

			data_1    = *(tInput + blockIdx.x*blockDim.x+ threadIdx.x - i);   //b(0)*x(n-0)+b(1)*x(n-1)+...+b(nb-2)*x(n-(nb-2))  

			data_sum += (data_1*param[i]);

		}

		*(tOutPut + blockIdx.x*blockDim.x + threadIdx.x) = data_sum;

	}


 


}




__global__ void Bandpass_back_1(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)
{
	

	float data_1;

	float data_sum;

	data_sum = 0.0;



	if (threadIdx.x <= iParaLen - 1)   {                               


		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);     

			data_sum += (data_1*param[i]);

		}


		data_1 = *(tInput + blockIdx.x * iWidth + iWidth - 1);                        //  x(N-1) 


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{

			data_sum += (data_1*param[j]);                                            // b(n+1)*x(N-1)+...+b(nb-1)*x(N-1) 

		}


		*(tOutPut + blockIdx.x * iWidth + iWidth - 1 - threadIdx.x) = data_sum;                 // y(n)

	}

	else    {                                                                             


		for (int i = 0; i <= iParaLen - 1; i++)
		{
			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);

			data_sum += (data_1*param[i]);                                         //  y(N-1-n) = b(0)*x(N-1-n+0) +b(1)*x(N-1-n+1)+...+b(nb-1)*x(N-1-n+nb-1)

		}

		*(tOutPut + blockIdx.x * iWidth + iWidth - 1 - threadIdx.x)   = data_sum;




	}



	__syncthreads();  

}






__global__ void Bandpass_back_1024(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)   {


	float  data_1;

	float  data_sum;

	data_sum = 0.0;


	int   line_serial;

	int    bid = blockIdx.x;


	line_serial = bid / 16;



	int  line_mod = bid % 16;










	if ((0 == line_mod))    {                                                                   


		if (threadIdx.x <= iParaLen - 1)   {                                                      


			for (int i = 0; i <= threadIdx.x; i++)
			{

				 
				data_1 = *(tInput + line_serial*iWidth + iWidth - 1 - threadIdx.x + i);     

				data_sum += (data_1*param[i]);

			}


			data_1 = *(tInput + line_serial * iWidth + iWidth - 1);                          // x(N-1) 


			for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
			{

				data_sum += (data_1*param[j]);                                                   // b(n+1)*x(N-1)+...+b(nb-1)*x(N-1) 

			}


			*(tOutPut + line_serial * iWidth + iWidth-1 - threadIdx.x) = data_sum;              // y(n)      

		}

		else  if (threadIdx.x  >iParaLen - 1)   {                                                 

			data_sum = 0;

			for (int i = 0; i <= iParaLen - 1; i++)
			{
				data_1 = *(tInput + line_serial*iWidth + iWidth - 1 - threadIdx.x + i);

				data_sum += (data_1*param[i]);                                               //  y(N-1-n) = b(0)*x(N-1-n+0) +b(1)*x(N-1-n+1)+...+b(nb-1)*x(N-1-n+nb-1)

			}

			*(tOutPut + line_serial * iWidth + iWidth - 1 - threadIdx.x) = data_sum;




		}



	}  

	else  {                                                                                     

		    data_sum = 0;
		  

		for (int i = 0; i <= iParaLen - 1; i++)
		{
			data_1 = *(tInput + line_serial*iWidth + iWidth - 1 - (threadIdx.x + line_mod*blockDim.x) + i);

			data_sum += (data_1*param[i]);                                               //  y(N-1-n) = b(0)*x(N-1-n+0) +b(1)*x(N-1-n+1)+...+b(nb-1)*x(N-1-n+nb-1)

		}

		*(tOutPut + line_serial * iWidth + iWidth - 1 - (threadIdx.x + line_mod*blockDim.x)) = data_sum;




	}
	
	
	
	
	}
















__device__      void   xcorr_cuda(const  Complex* templateMat_startID, const Complex* objectMat_startID, Complex*resultMat_startID)     {


         #pragma unroll

	for (int i = 0; i < 101; i++)   {


		Complex     sum_object = 0;

		Complex     frac_object = 0;


		Complex     pow_template = 0;


		Complex      pow_object = 0;


		Complex     result = 0;


	

		for (int j = 0; j < 100; j++)  {


			sum_object += *(objectMat_startID + i + j);


		}

		

		Complex   ave_object =   sum_object / 100;


		

		for (int j = 0; j < 100; j++)  {

			Complex    tmp = *(templateMat_startID + j) *  (*(objectMat_startID + i + j) - ave_object);


			frac_object += tmp;

		}


	

		for (int j = 0; j < 100; j++)  {


			pow_template += *(templateMat_startID + j) * *(templateMat_startID + j);

		}

	

		for (int j = 0; j < 100; j++)  {


			pow_object += *(objectMat_startID + i + j)* * (objectMat_startID + i + j);

		}

		

		result = sqrt(pow_template*pow_object);

		

		*(resultMat_startID + i) = frac_object / result;

	}


}



__device__      void   minMax_cuda(Complex*resultMat_startID, Complex* min_value, Complex*  max_value, int * max_location)   {



	for (int i = 0; i < 101; i++)  {

		if (*(resultMat_startID + i) >= *max_value)  {

			*max_location   = i;

			*max_value = *(resultMat_startID + i);



		}

	}




}


 


__device__    void    interp_cuda(Complex*resultMat_startID, int *  max_loc, Complex*max_value, int * multiWin, int * winSize, Complex*  displace)     {

	Complex*pre = (Complex*)resultMat_startID + *max_loc - 1;

	Complex*next = (Complex*)resultMat_startID + *max_loc + 1;


	*displace   = (*multiWin - 1) * *winSize / 2 - *max_loc - (*pre - *next) / (2 * (*pre - 2 * *max_value + *next));


}






__global__   void  displacement_api_cuda(Complex*disInputCuda, int rows, int cols, int  multiWin, int winSize, int  stepSize, templateMat*templateMatShare, objectMat* objectMatShare, resultMat*resultMatShare, Complex*min, Complex*max, int*max_location, Complex* displacement )      {


	int   out_offset = blockIdx.x *blockDim.x + threadIdx.x;                     

	int    bid       = blockIdx.x ;                                              
	
	int    tid       = threadIdx.x;                                        




	    Complex*templateMatID;                               //ID


	    Complex*objectMatID;                                //ID




	      templateMatID = (Complex*)(disInputCuda + blockIdx.x*cols + (multiWin - 1) * winSize / 2 + threadIdx.x * stepSize);




		  for (int i = 0; i < 100;i++)  {

			  if (i < 64)    {
			  
				  templateMatShare[out_offset].tempData.elem[i]= *(templateMatID + i);
			  
			  }
		  
			  else
				      

			    templateMatShare[out_offset].tempData.atom[i-64] = *(templateMatID + i);
		  
		  
		  }



		  objectMatID   = (Complex*)(disInputCuda + (blockIdx.x + 1)*cols + threadIdx.x * stepSize);


   
		  for (int i = 0; i < 200; i++)  {

			  if (i<64)
				  objectMatShare[out_offset].objData.elem_0[i]     = *(objectMatID + i);
			  else if (i<128)
				  objectMatShare[out_offset].objData.elem_1[i - 64] = *(objectMatID + i);

			  else if (i<192)
				  objectMatShare[out_offset].objData.elem_2[i - 128] = *(objectMatID + i);
			  else
				  objectMatShare[out_offset].objData.atom[i - 192]   = *(objectMatID + i);

			

		  }

       



		  for (int i = 0; i < 101; i++)  {
		   
			  if (i<64)
				  resultMatShare[out_offset].resData.elem[i]   = 0;
		   
			  else
				  resultMatShare[out_offset].resData.atom[i-64] = 0;
		  
		  }
		 





		  xcorr_cuda(templateMatShare[out_offset].tempData.elem, objectMatShare[out_offset].objData.elem_0, resultMatShare[out_offset].resData.elem);



		minMax_cuda(resultMatShare[out_offset].resData.elem, &min[out_offset], &max[out_offset], &max_location[out_offset]);


		interp_cuda(resultMatShare[out_offset].resData.elem, &max_location[out_offset], &max[out_offset], &multiWin, &winSize, &displacement[out_offset]);


	

}





__global__  void   remove_singular_cuda(Complex*disOutputCuda, Complex*singularOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                                           

	int    bid   = blockIdx.x;                                                                     //  block   id

	int    tid   = threadIdx.x;                                                                    //  thread  id    

	int    offrow = 0;

	if (bid  > 0 && bid < gridDim.x - 1 && tid < blockDim.x-1 )   {
	
		    offrow = (blockIdx.x - 1)*blockDim.x + threadIdx.x;
	
	} 




	if (bid > 0 && bid < gridDim.x - 1 && tid < blockDim.x - 1 && (abs(disOutputCuda[offset]) > 12))  {

		singularOutputCuda[offset] = disOutputCuda[offrow];

	}

	else  {

		singularOutputCuda[offset] = disOutputCuda[offset];

	}


}



__global__   void   displace_add_cuda(Complex*singularOutputCuda, Complex*addOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                               

	int    bid   = blockIdx.x;                                                          // block   id

	int    tid   = threadIdx.x;                                                         // thread  id   

	int   offrow = (bid >0 ) ? ( (blockIdx.x - 1)*blockDim.x + threadIdx.x)  :0 ;       

	int   nextoff =  (blockIdx.x + 1)*blockDim.x + threadIdx.x;


	Complex  sum = 0.0;




	if (bid > 0)  {

		     

		for (int i = 0; i < bid; i++)   {

			int  off = i*blockDim.x + threadIdx.x;


			sum = sum + singularOutputCuda[off];

		}


		addOutputCuda[offset] = singularOutputCuda[offset] + sum;


	}

	else   {

		addOutputCuda[offset] = singularOutputCuda[offset];

	}













}





__global__   void   extend_data_cuda(Complex*addOutputCuda, Complex*extendOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                        

	int    bid   = blockIdx.x;                                                  // block   id

	int    tid   = threadIdx.x;                                                 // thread  id   


	if (tid<N - 1)  {

		int   add_base = blockIdx.x *(blockDim.x - (N - 1));

		extendOutputCuda[offset] = addOutputCuda[add_base];                    //  extend  primites

	}

	else
	{

		int   extoff = blockIdx.x *(blockDim.x - (N - 1)) + threadIdx.x - (N - 1);

		extendOutputCuda[offset] = addOutputCuda[extoff];

	}

}


  

__global__ void  smooth_filter_cuda(Complex*extendOutputCuda, Complex* smoothOutputCuda)   {

	int   offset  = blockIdx.x *blockDim.x + threadIdx.x;

	int   extbase = blockIdx.x*(blockDim.x + N - 1) + threadIdx.x;              

	int    bid = blockIdx.x;                                                    // block   id

	int    tid = threadIdx.x;                                                   // thread  id  


	Complex   sum = 0;


	for (int i = extbase; i < extbase + N; i++)  {


		Complex  temp = *(extendOutputCuda + i);

		sum = sum + temp;


	}

	smoothOutputCuda[offset] = sum / N;

}




__global__  void   timeField_filter_cuda(const Complex* smoothOutputCuda, const float* param,  const int  steps, Complex* timeFilterOutputCuda)    {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                       


	int    bid = blockIdx.x;                                                    // block   id

	int    tid = threadIdx.x;                                                   // thread  id     

	Complex  sum_temp = 0;

	float    coeff   = 0;

	for (int i = 0; i <= bid; i++)   {

		if ((bid - i) < steps)

			coeff = param[bid - i];

		else

			coeff = param[0];


		sum_temp += smoothOutputCuda[i*blockDim.x + threadIdx.x] * coeff;


	}

	timeFilterOutputCuda[offset] = sum_temp;

}






bool    CudaMain::isAvailable()  {

	int   count = 0;

	printf("Start to detecte devices.........\n");                   

	hipGetDeviceCount(&count);                                    

	if (count == 0){

		fprintf(stderr, "There is no device.\n");

		return false;

	}


	printf("%d device/s detected.\n", count);                      


	int i;

	for (i = 0; i < count; i++){                                 

		hipDeviceProp_t prop;

		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {  

			if (prop.major >= 1)                                

			{
				printf("Device %d: %s supports CUDA %d.%d.\n", i + 1, prop.name, prop.major, prop.minor);
				break;


			}
		}
	}

	if (i == count) {                                         
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);                                       

	return true;

}





CudaMain::CudaMain()  {

	


	cpu_inputMat      = NULL;
	
	cpu_SplineOutMat  = NULL ;

	cpu_RadonMat      = NULL;

	cpu_WaveRate      = 0  ;

	mallocFlag        = false;

	cpu_config        = new   ConfigParam ;

	cpu_disMat        = NULL;








	inputMat         = NULL;

	zeroFilterMat    = NULL;

	frontFilterMat   = NULL;

	disOutput        = NULL;

	bandfilterParam  = NULL;

	lowfilterParam   = NULL;

	matchfilterParam = NULL;

	lowFrontMat      = NULL;

	lowBackMat       = NULL;

	singularOutputCuda = NULL;

	addOutputCuda      = NULL;

	extendOutputCuda   = NULL;


	radonIn          = NULL;

	radonOut         = NULL;


}




CudaMain :: ~CudaMain()  {

	freeMem(); 



}



void   CudaMain::inputConfigParam( ConfigParam*config) {



	cpu_config = config;


}



void  CudaMain::inputRfData(const EInput& in) {    

	float* input = in.pDatas;


	for (int i = 0; i < cpu_inputMat->rows; i++)
	{
		for (int j = 0; j < cpu_inputMat->cols; j++)
		{
			*(static_cast<float*>(static_cast<void*>(CV_MAT_ELEM_PTR(*cpu_inputMat, i, j)))) = input[i * cpu_inputMat->cols + j];
		}
	}


}




void  CudaMain::getbandFilterParam(std::string paramFileName) {

	if (paramFileName.size() == 0)
	{
		exit(1);
	}

	std::fstream paramFile(paramFileName.c_str());

	if (!paramFile)
	{
		exit(1);
	}

	float tmp;

	std::string str;


	cpu_bandfilterParam.clear();

	while (!paramFile.eof())
	{
		paramFile >> tmp;
		cpu_bandfilterParam.push_back(tmp);
	}
	paramFile.close();



}


void   CudaMain::getlowFilterParam(std::string paramFileName) {

	if (paramFileName.size() == 0)
	{
		exit(1);
	}

	std::fstream paramFile(paramFileName.c_str());

	if (!paramFile)
	{
		exit(1);
	}

	float tmp;

	std::string str;


	cpu_lowfilterParam.clear();

	while (!paramFile.eof())
	{
		paramFile >> tmp;
		cpu_lowfilterParam.push_back(tmp);
	}
	paramFile.close();


}



void  CudaMain::getmatchFilterParam(std::string paramFileName) {

	if (paramFileName.size() == 0)
	{
		exit(1);
	}

	std::fstream paramFile(paramFileName.c_str());

	if (!paramFile)
	{
		exit(1);
	}

	float tmp;

	std::string str;


	cpu_matchfilterParam.clear();

	while (!paramFile.eof())
	{
		paramFile >> tmp;
		cpu_matchfilterParam.push_back(tmp);
	}
	paramFile.close();

}











void  CudaMain::mallocMem(void)  {

	mallocMats();

	mallocGPUMem();

	

}



void CudaMain::freeMem(void)  {

	freeMats();
   
	deleteGPUMem();
}





void   CudaMain::mallocGPUMem() {



	int  MatRows = cpu_config->shearFrameLineNum;

	int  MatCols = cpu_config->sampleNumPerLine ;

	int windowHW = cpu_config->windowHW;

	int maxLag   = cpu_config->maxLag;

	int step     = cpu_config->step;


	int interpnum  = cpu_config->fitline_pts;

	int iBPParaLen = 40;                                                     

	iBPParaLen     = (iBPParaLen > cpu_bandfilterParam.size()) ? iBPParaLen : cpu_bandfilterParam.size();


	int iLPParaLen = 40;                                                    

	iLPParaLen     = (iBPParaLen > cpu_lowfilterParam.size()) ? iBPParaLen : cpu_lowfilterParam.size();


	int iMHParaLen = 40;                                                     

	iMHParaLen    = (iBPParaLen > cpu_matchfilterParam.size()) ? iBPParaLen : cpu_matchfilterParam.size();






	if (MatRows == 0 || MatCols == 0)
	{

		printf("  row  and col  is zero! call InputConfigParas first!\n");
		return;

	}

	hipError_t cudaStatus = hipSetDevice(0);                                

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}



	int  multiWin = 2;                                                      //  �󴰿ڶ�С���ڵı���

	int cxorrLines = MatRows - 1;                                           //  λ�ƾ������ɨ������Ŀ        299

	int iOutRows = (MatCols - multiWin*windowHW) / step;                    //  λ�ƾ��������Ҫƥ��Ķ���     799 

	int extRows = iOutRows + N - 1;                                         //  ��չ����  799+100-1

	hipMalloc(&disOutput, cxorrLines *iOutRows* sizeof(Complex));          //  λ�ƾ���GPU�ڴ����







	hipMalloc(&singularOutputCuda, cxorrLines *iOutRows* sizeof(Complex)); // ȥ���������GPU�ڴ����


	hipMalloc(&addOutputCuda, cxorrLines *iOutRows* sizeof(Complex));      // λ�Ƶ�����GPU�ڴ����


	hipMalloc(&extendOutputCuda, cxorrLines *extRows* sizeof(Complex));     // ��չ������GPU�ڴ����



	hipMalloc(&inputMat, MatRows * MatCols * sizeof(Complex));             //   ���������GPU�϶�Ӧ���ڴ棻


	hipMalloc(&zeroFilterMat, MatRows * MatCols * sizeof(Complex));       //   ��ͨ����λ�˲������GPU�ڴ���䣻


	hipMalloc(&frontFilterMat, MatRows * MatCols * sizeof(Complex));     //  ��ͨ����λ���˲���GPU�ڴ����



	hipMalloc(&lowBackMat, cxorrLines * iOutRows * sizeof(Complex));       //   ��ͨ����λ�˲������GPU�ڴ���䣻


	hipMalloc(&lowFrontMat, cxorrLines * iOutRows * sizeof(Complex));     //  ��ͨ����λ���˲���GPU�ڴ����

	


	hipMalloc(&bandfilterParam, iBPParaLen * sizeof(float));                // iBPParaLen�˲�������40


	hipMalloc(&lowfilterParam, iLPParaLen * sizeof(float));                // iLPParaLen�˲�������40


	hipMalloc(&matchfilterParam, iMHParaLen * sizeof(float));              // iMHParaLen�˲�������40



	hipMalloc(&fit_IN, cxorrLines *iOutRows* sizeof(Complex));          //  λ�ƾ���GPU�ڴ����


	int   points = 5;


	int   strain_col = iOutRows - points + 1;

	hipMalloc(&fit_Out, cxorrLines *strain_col* sizeof(Complex));          //  λ�ƾ���GPU�ڴ����






	int RadonInputCols      = 1961;                                    

	int RadonInputRows      = 4;                                       

	hipMalloc(&radonIn, sizeof(float) * RadonInputCols * RadonInputRows);                //�����任GPU����

	hipMalloc(&radonOut, sizeof(float) * RadonInputCols * (RadonInputCols - 1));        //�����任GPU���  


	mallocFlag             = true;




}





void  CudaMain::deleteGPUMem()  {


	if (inputMat != NULL)
	{
		hipFree(inputMat);

		inputMat = NULL;
	}

	
	if (zeroFilterMat != NULL)
	{
		hipFree(zeroFilterMat);
		zeroFilterMat = NULL;
	}


	if (frontFilterMat != NULL)
	{
		hipFree(frontFilterMat);
		frontFilterMat = NULL;
	}


	if (lowBackMat != NULL)
	{
		hipFree(lowBackMat);
		lowBackMat = NULL;
	}


	if (lowFrontMat != NULL)
	{
		hipFree(lowFrontMat);
		lowFrontMat = NULL;
	}




	if (disOutput != NULL)
	{
		hipFree(disOutput);
		disOutput = NULL;
	}




	if (singularOutputCuda != NULL)
	{
		hipFree(singularOutputCuda);

		singularOutputCuda = NULL;
	}


	if (addOutputCuda != NULL)
	{
		hipFree(addOutputCuda);

		addOutputCuda = NULL;
	}


	if (extendOutputCuda != NULL)
	{
		hipFree(extendOutputCuda);

		extendOutputCuda = NULL;
	}







	if (bandfilterParam != NULL)
	{
		hipFree(bandfilterParam);
		bandfilterParam = NULL;
	}

	
	if (lowfilterParam != NULL)
	{
		hipFree(lowfilterParam);
		lowfilterParam = NULL;
	}



	if (matchfilterParam != NULL)
	{
		hipFree(matchfilterParam);
		matchfilterParam = NULL;
	}






	if (radonIn != NULL)
	{
		hipFree(radonIn);
	}

	if (radonIn != NULL)
	{
		hipFree(radonIn);
	}


	hipDeviceReset();

	mallocFlag = false;


}




void  CudaMain::mallocMats() {


	cpu_inputMat    =   cvCreateMat(cpu_config->shearFrameLineNum, cpu_config->sampleNumPerLine, CV_32FC1);         //�������

	int  MatRows    = cpu_config->shearFrameLineNum;

	int  MatCols    = cpu_config->sampleNumPerLine;

	int windowHW    = cpu_config->windowHW;

	int maxLag      = cpu_config->maxLag;

	int step        = cpu_config->step;


	int  multiWin   = 2;                                                    //  �󴰿ڶ�С���ڵı���

	int cxorrLines  = MatRows - 1;                                         //   λ�ƾ������ɨ������Ŀ        299

	int iOutRows    = (MatCols - multiWin*windowHW) / step;               //    λ�ƾ��������Ҫƥ��Ķ���     799 

	cpu_disMat      = cvCreateMat(cxorrLines, iOutRows, CV_32FC1);       //     λ�ƾ���   



	int  fit_point  = 5;

	
	int  fit_cols = iOutRows - fit_point + 1;

	cpu_fitMat = cvCreateMat(cxorrLines, fit_cols, CV_32FC1);                 





	cpu_SplineOutMat = cvCreateMat(1962, 4, CV_32FC1);                  //    SplineOutMat��������ڻ�ͼ���ȽϽ��  

		
	cpu_RadonMat    = cvCreateMat(1962, 4, CV_32FC1);                  //     radon������Ƚϼ�����  



	mallocFlag     = false; 
	


}



void   CudaMain::freeMats() {

	if (cpu_inputMat != NULL)
	{
		cvReleaseMat(&cpu_inputMat);
		cpu_inputMat = NULL;
	}
	

	if (cpu_disMat != NULL)
	{
		cvReleaseMat(&cpu_disMat);
		cpu_disMat = NULL;
	}
	

	if (cpu_SplineOutMat != NULL)
	{
		cvReleaseMat(&cpu_SplineOutMat);
		cpu_SplineOutMat = NULL;
	}

	if (cpu_RadonMat != NULL)
	{
		cvReleaseMat(&cpu_RadonMat);
		cpu_RadonMat = NULL;
	}
	

	mallocFlag = NULL;


	free(cpu_config);


	cpu_config = NULL;

	  
}





CvMat*  CudaMain::bandpassFilt_cuda(CvMat* rawMat)  {


	Complex* h_MatData = (Complex*)rawMat->data.fl;

	hipMemsetAsync(frontFilterMat, 0, sizeof(Complex)*rawMat->cols*rawMat->rows);

	hipMemcpyAsync(zeroFilterMat, h_MatData, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyHostToDevice);    

	int steps = cpu_bandfilterParam.size();

	hipMemcpyAsync(bandfilterParam, &cpu_bandfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);                 





	dim3 blockID, threadID;

	blockID.x  = rawMat->rows;

	threadID.x = rawMat->cols;

	hipDeviceSynchronize();

	Bandpass_front_1 <<<blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);

	hipDeviceSynchronize();


	hipMemcpy(zeroFilterMat, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToDevice);


	Bandpass_back_1 << <blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);


	hipDeviceSynchronize();

	   
	hipFree(bandfilterParam);

	hipMemcpy(h_MatData, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToHost);   

	hipFree(zeroFilterMat);

	hipFree(frontFilterMat);


	return rawMat;


}





CvMat*  CudaMain::bandpassFilt_1024_cuda(CvMat* rawMat)  {


	Complex* h_MatData = (Complex*)rawMat->data.fl;

	hipMemsetAsync(frontFilterMat, 0, sizeof(Complex)*rawMat->cols*rawMat->rows);

	hipMemcpyAsync(zeroFilterMat, h_MatData, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyHostToDevice);    

	int steps = cpu_bandfilterParam.size();

	hipMemcpyAsync(bandfilterParam, &cpu_bandfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);                 


	dim3 blockID, threadID;

	blockID.x = rawMat->rows*16;                           





	threadID.x = rawMat->cols/16;                      



	hipDeviceSynchronize();

	Bandpass_front_1024 << <blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);

	hipDeviceSynchronize();



	hipMemcpy(zeroFilterMat, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToDevice);


	Bandpass_back_1024 << <blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);



	hipDeviceSynchronize();


	hipFree(bandfilterParam);

	hipMemcpy(h_MatData, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToHost);   

	hipFree(zeroFilterMat);

	hipFree(frontFilterMat);



	return rawMat;
 






}










void  CudaMain::zeroFilter_cuda(CvMat* rawMat, Complex*filterOutput) {







}





void   CudaMain::computeDisplacement_cuda(CvMat* filtOutMat, int  multiWin, int winSize, int stepSize, CvMat*outputMat){



	int     WinNum    = (filtOutMat->cols - multiWin*winSize) / stepSize;     

	Complex* hInput   = (Complex*)filtOutMat->data.fl;                        

	Complex*hOutput  = (Complex*)outputMat->data.fl;                        


	hipMemcpy(inputMat, hInput, filtOutMat->cols*filtOutMat->rows*sizeof(Complex), hipMemcpyHostToDevice);   

	dim3 dBlock;

	dim3 dThread;

	dBlock.x = filtOutMat->rows - 1;                             


	dThread.x = WinNum;                                           


	templateMat*templateMatShare;                               


	objectMat* objectMatShare;                                  



	resultMat*resultMatShare;                                  



	Complex*      min;


	Complex*      max;

	int*          max_location;


	Complex*      displacement;



	hipMalloc(&templateMatShare, dBlock.x*dThread.x* sizeof(templateMat));             


	hipMalloc(&objectMatShare,  dBlock.x*dThread.x* sizeof(objectMat));               


	hipMalloc(&resultMatShare,  dBlock.x*dThread.x* sizeof(resultMat));            



	hipMalloc(&min, dBlock.x*dThread.x* sizeof(Complex));                          


	hipMalloc(&max, dBlock.x*dThread.x* sizeof(Complex));                         


	hipMalloc(&max_location, dBlock.x*dThread.x* sizeof(int));                     


	hipMalloc(&displacement, dBlock.x*dThread.x* sizeof(Complex));               



	displacement_api_cuda << < dBlock, dThread >> >   (inputMat, filtOutMat->rows, filtOutMat->cols, multiWin, winSize, stepSize, templateMatShare, objectMatShare, resultMatShare, min, max, max_location, displacement);

	hipDeviceSynchronize();


	hipFree(templateMatShare);

	hipFree(objectMatShare);

	hipFree(resultMatShare);

	hipFree(min);

	hipFree(max);

	hipFree(max_location);

                               

	remove_singular_cuda << <dBlock, dThread >> >   (displacement, singularOutputCuda);

	hipDeviceSynchronize();

               

	displace_add_cuda << <dBlock, dThread >> >  (singularOutputCuda, addOutputCuda);

	hipDeviceSynchronize();
   

	int  ext_threads = dThread.x + N - 1;

	extend_data_cuda << < dBlock, ext_threads >> > (addOutputCuda, extendOutputCuda);

	hipDeviceSynchronize();

	hipFree(addOutputCuda);
 

	smooth_filter_cuda << <dBlock, dThread >> >   (extendOutputCuda, disOutput);

	hipDeviceSynchronize();

	hipFree(extendOutputCuda);


	int steps = cpu_matchfilterParam.size();

	hipMemcpyAsync(matchfilterParam, &cpu_matchfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);             


	timeField_filter_cuda << <dBlock, dThread >> > (disOutput, matchfilterParam,  steps, singularOutputCuda);

	hipDeviceSynchronize();

	hipFree(disOutput);


	hipMemcpy(hOutput, singularOutputCuda, dBlock.x  * dThread.x*sizeof(Complex), hipMemcpyDeviceToHost);   


	hipFree(singularOutputCuda);




}






void   CudaMain::zeroDisplacement_cuda(CvMat* inputMat, int  multiWin, int winSize, int stepSize, Complex*disOutput){





}




__global__ void     lowpass_front_799(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)    {


	float data_sum;

	float data_1;

	data_sum = 0.0;


	if (threadIdx.x <= iParaLen - 1)                                   
	{

		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);      //b(0)*x(n-0)+b(1)*x(n-1)+...+b(n)*x(0)   

			data_sum += (data_1*param[i]);

		}

		data_1 = *(tInput + blockIdx.x * iWidth);                          // x(0)


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{
			data_sum += (data_1*param[j]);                                 //b(n+1)*x(0)+...+b(nb-2)*x(0)
		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x) = data_sum;



	}
	else                                                                         
	{
		
		for (int i = 0; i <= iParaLen - 1; i++)
		{

			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);   //b(0)*x(n-0)+b(1)*x(n-1)+...+b(nb-2)*x(n-(nb-2))  

			data_sum += (data_1*param[i]);

		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x) = data_sum;

	}




}




__global__  void   lowpass_back_799(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)   {


	

	float data_1;

	float data_sum;

	data_sum = 0.0;



	if (threadIdx.x <= iParaLen - 1)   {                              


		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);      //

			data_sum += (data_1*param[i]);

		}


		data_1 = *(tInput + blockIdx.x * iWidth + iWidth - 1);                        //  x(N-1) 


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{

			data_sum += (data_1*param[j]);                                            // b(n+1)*x(N-1)+...+b(nb-1)*x(N-1) 

		}


		*(tOutPut + blockIdx.x * iWidth + iWidth - 1 -threadIdx.x) = data_sum;                 // y(n)

	}

	else    {                                                                             


		for (int i = 0; i <= iParaLen - 1; i++)
		{
			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);

			data_sum += (data_1*param[i]);                                         //  y(N-1-n) = b(0)*x(N-1-n+0) +b(1)*x(N-1-n+1)+...+b(nb-1)*x(N-1-n+nb-1)

		}

		*(tOutPut + blockIdx.x * iWidth + iWidth - 1 - threadIdx.x) = data_sum;




	}



}









CvMat*  CudaMain::lowpassFilt_799_cuda(CvMat* disMat)  { 


	Complex* h_MatData = (Complex*)disMat->data.fl;

	hipMemsetAsync(lowBackMat, 0, sizeof(Complex)*disMat->rows*disMat->cols);

	hipMemcpyAsync(lowFrontMat, h_MatData, sizeof(Complex)*disMat->rows*disMat->cols, hipMemcpyHostToDevice);          

	int steps = cpu_lowfilterParam.size();

	hipMemcpyAsync(lowfilterParam, &cpu_lowfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);                





	dim3 blockID, threadID;

	blockID.x = disMat->rows;

	threadID.x = disMat->cols;


	lowpass_front_799 << <blockID, threadID >> >(lowFrontMat, disMat->cols, lowfilterParam, steps, lowBackMat);

	hipDeviceSynchronize();

	
	hipMemcpy(lowFrontMat, lowBackMat, sizeof(Complex)*disMat->rows*disMat->cols, hipMemcpyDeviceToDevice);


	lowpass_back_799 << <blockID, threadID >> >(lowFrontMat, disMat->cols, lowfilterParam, steps, lowBackMat);


	hipDeviceSynchronize();


	hipFree(lowfilterParam);

	hipMemcpy(h_MatData, lowBackMat, sizeof(Complex)*disMat->cols*disMat->rows, hipMemcpyDeviceToHost);   

	hipFree(lowFrontMat);

	hipFree(lowBackMat);


	return disMat;




}


__device__  void    fitLine_cv_func(Complex*xx_tmp, Complex*yy_tmp, Complex* result)   {


	Complex xmean = 0.0f;

	Complex ymean = 0.0f;

	for (int i = 0; i < 5; i++)
	{
		xmean += xx_tmp[i];

		ymean += yy_tmp[i];

	}


	xmean /= 5;

	ymean /= 5;


	Complex sumx2 = 0.0f;

	Complex sumxy = 0.0f;

	for (int i = 0; i < 5; i++)
	{

		sumx2 += (xx_tmp[i] - xmean) * (xx_tmp[i] - xmean);

		sumxy += (yy_tmp[i] - ymean) * (xx_tmp[i] - xmean);

	}


	*result = (Complex)(sumxy / sumx2);

}




__global__  void  fitLine_L2_cuda(Complex*strain_IN, Complex*xx_IN,  Complex*strainOut)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                       


	int    bid = blockIdx.x;                                                    // block   id

	int    tid = threadIdx.x;                                                   // thread  id     


	int    act_off = blockIdx.x *(blockDim.x + 5 - 1) + threadIdx.x;  // input



	Complex   xx_tmp[5];

	Complex  yy_tmp[5];


	for (int i = 0; i < 5; i++)  {

		xx_tmp[i] = xx_IN[act_off + i];

		yy_tmp[i] = strain_IN[act_off + i];


	}


	fitLine_cv_func(xx_tmp, yy_tmp, &strainOut[offset]);


}







void  CudaMain::strainCalculate_cuda(CvMat*disMat,    CvMat* fitMat)  {


	Complex* h_MatData = (Complex*)disMat->data.fl;


	Complex* out_MatData = (Complex*)fitMat->data.fl;


	hipMemcpyAsync(fit_IN, h_MatData, sizeof(Complex)*disMat->rows*disMat->cols, hipMemcpyHostToDevice);          


	int   fit_points = 5;


	dim3 blockID, threadID;

	blockID.x = disMat->rows;

	threadID.x = disMat->cols - fit_points +1;




	int   xx_rows = disMat->rows;

	int  xx_cols  = disMat->cols;


	CvMat*    xx_mat = cvCreateMat(xx_rows, xx_cols, CV_32FC1);


	   
	for (int i = 0; i < xx_rows; i++)   {

		for (int j = 0; j < xx_cols; j++)  {
		
		
			*(static_cast<float*>(static_cast<void*>(CV_MAT_ELEM_PTR(*xx_mat, i, j)))) = j*fit_points;
				
		}
	
	}


	Complex* xx_IN  ;

    hipMalloc(&xx_IN, sizeof(Complex)*xx_rows*xx_cols);           


	Complex* xx_Data = (Complex*)xx_mat->data.fl;


	hipMemcpyAsync(xx_IN, xx_Data, sizeof(Complex)*xx_mat->rows*xx_mat->cols, hipMemcpyHostToDevice);



	fitLine_L2_cuda << <blockID, threadID >> >  (fit_IN, xx_IN,  fit_Out);


	hipMemcpy(out_MatData, fit_Out, sizeof(Complex)*fitMat->rows*fitMat->cols, hipMemcpyDeviceToHost);   




	hipFree(fit_IN);

	hipFree(xx_IN);

	hipFree(fit_Out);







}




void  CudaMain::ImagePostProc(IplImage *strImage, const char *filename, const CvPoint &start, const CvPoint &end)
{

	const char * gray_file = "strain_gpu_gray.bmp";


	{
		IplImage *pimgStrain = cvCreateImage(cvGetSize(strImage), strImage->depth, 3);

		cvCvtColor(strImage, pimgStrain, CV_GRAY2BGR);

		cvSaveImage(gray_file, pimgStrain);

		cvReleaseImage(&pimgStrain);

	}

	{

		IplImage *pImage = cvLoadImage(gray_file, 0);

		IplImage *pimgStrain = cvCreateImage(cvGetSize(pImage), pImage->depth, 3);

		pimgStrain = cvCreateImage(cvGetSize(pImage), pImage->depth, 3);


	
		ImageAdjust(pImage, pImage, 0, 0.5, 0, 0.5, 0.6);

		

		cvNot(pImage, pImage);

		
		cvCvtColor(pImage, pimgStrain, CV_GRAY2BGR);


		ChangeImgColor(pimgStrain);


		cvLine(pimgStrain, start, end, CV_RGB(255, 0, 0), 2, CV_AA, 0);  


		cvSaveImage(filename, pimgStrain);


		
		cvReleaseImage(&pImage);

		cvReleaseImage(&pimgStrain);

	}

}






//////////////////////////////////////////////////////////////////////////
// �����任
// pmatDisplacement,   rows: disp;  cols: time-extent( lines)
//     ��,��ʾһ����, Ҳ����ʱ�� ��
//     ��,��ʾӦ���ֵ
//////////////////////////////////////////////////////////////////////////

void   CudaMain::RadonSum(const CvMat *pmatDisplacement, CvMat **ppmatRodan) {


	int xstart          = 0;

	int xend            = pmatDisplacement->rows;                     

	int t               = pmatDisplacement->cols;                    

	CvMat *pmatRodan    = cvCreateMat(t - 1, t, pmatDisplacement->type);

	cvZero(pmatRodan);

	int tstart          = 0;

	int tend            = 0;

	int dx              = 0;

	float dt            = 0.0f;

	float c             = 0.0f;


	for (tstart = 0; tstart < t - 1; tstart++)
	{

		for (tend = tstart + 1; tend < t; tend++)
		{

			c = (float)(xend - xstart) / (tend - tstart);                     //k

			for (dx = xstart; dx < xend; dx++)
			{

				dt = tstart + (dx - xstart) / c;                             //

				CV_MAT_ELEM(*pmatRodan, float, tstart, tend) = CV_MAT_ELEM(*pmatRodan, float, tstart, tend)
					+ CV_MAT_ELEM(*pmatDisplacement, float, dx, (int)dt);

			}
		}
	}


	*ppmatRodan = pmatRodan;






}






void  CudaMain::RadonProcess2(CvPoint &s, CvPoint &e, ConfigParam*config, const CvRect &sub_rc, const CvMat &matStrain)
{

	int  radon_num = config->radon_num;                    
	 

	int  radon_step = config->radon_step;                  



	int  intpl_multiple = 1;                                



	std::vector<RadonParam> array_params;



	for (int i = 0; i < radon_num; i++)                     
	{


		RadonParam param;

		param.rc.x = sub_rc.x;

		param.rc.y = sub_rc.y + i*radon_step;

		param.rc.width = sub_rc.width;

		param.rc.height = sub_rc.height;


		CvMat *pmatSub = cvCreateMatHeader(param.rc.height-1, param.rc.width-1, matStrain.type);


		cvGetSubRect(&matStrain, pmatSub, cvRect(param.rc.x, param.rc.y, param.rc.width-1, param.rc.height-1));


		CvMat *pmatRadon = 0;


		CvMat *pmatMultiple = cvCreateMat(pmatSub->rows, pmatSub->cols * intpl_multiple, pmatSub->type);


		cvResize(pmatSub, pmatMultiple);


		RadonSum(pmatMultiple, &pmatRadon);


		double  min_val;


		double  max_val;


		CvPoint min_loc;


		CvPoint max_loc;


		cvMinMaxLoc(pmatRadon, &min_val, &max_val, &min_loc, &max_loc);


		param.pt = max_loc;


		param.xWidth = param.pt.y - param.pt.x;//add by wxm


		array_params.push_back(param);


		cvReleaseMat(&pmatRadon);


		cvReleaseMat(&pmatMultiple);


		cvReleaseMatHeader(&pmatSub);


	}


	std::sort(array_params.begin(), array_params.end(), MyLessThan2());



	if (config->calc_type.compare("middle") == 0)
	{

		int size = array_params.size();


		s.x = array_params[size / 2].pt.y / intpl_multiple;


		s.y = array_params[size / 2].rc.y;


		e.x = array_params[size / 2].pt.x / intpl_multiple;


		e.y = array_params[size / 2].rc.y + array_params[size / 2].rc.height-1;

	}

	else if (config->calc_type.compare("max") == 0)
	{

		int size = array_params.size();

		s.x = array_params[0].pt.y / intpl_multiple;

		s.y = array_params[0].rc.y;


		e.x = array_params[0].pt.x / intpl_multiple;

		e.y = array_params[0].rc.y + array_params[0].rc.height-1;

	}

	else if (config->calc_type.compare("min") == 0)

	{

		int size = array_params.size();

		s.x = array_params[size - 1].pt.y / intpl_multiple;

		s.y = array_params[size - 1].rc.y;


		e.x = array_params[size - 1].pt.x / intpl_multiple;

		e.y = array_params[size - 1].rc.y + array_params[size - 1].rc.height-1;

	}

	else
	{
		//

	}










}








void    CudaMain::random_proess_cuda(CvMat*fitMat, ConfigParam*config, EOutput &output)  {

	
	

		int    win_size          = config->windowHW;                                             


		double overlap           = (config->windowHW - config->step) / (float)config->windowHW;  

		double sound_velocity    = config->acousVel;                                            


		double sample_frq        = config->sampleFreqs;                                                             

		double prf               = 1 / 300e-6;                                                  


		int    dep_start         = (config->sb_x < 0) ? 0 : config->sb_x;

		int    dep_size          = (config->sb_w < 0) ? fitMat->width : config->sb_w;

		int    dep_end           = dep_start + dep_size - 1;

		int    t_start           = (config->sb_y < 0) ? 0 : config->sb_y;

		int    t_size            = (config->sb_h < 0) ? fitMat->rows : config->sb_h;

		int    t_end             = t_start + t_size - 1;


		CvMat *pmatStrainTran    = cvCreateMat(fitMat->cols, fitMat->rows, fitMat->type);     


		cvTranspose(fitMat, pmatStrainTran);


		CvPoint                   start;

		CvPoint                    end;
		
		CvRect                     rect;


		rect.x                    = t_start;

		rect.y                    = dep_start;

		rect.width                = t_size;

		rect.height               = dep_size;



		
#if 1
		RadonProcess2(start, end, config ,rect, *pmatStrainTran);
#endif






		double v                  = ((end.y - start.y) * win_size * (1 - overlap) * sound_velocity / sample_frq / 2)
			/ ((end.x - start.x) / prf);



		double e                  = v * v * 3;


		output.v                  = (float)v;


		output.e                  = (float)e;

		cvReleaseMat(&pmatStrainTran);


		
    

}












void  CudaMain::process(const EInput &input, EOutput& output) {




	   bandpassFilt_1024_cuda(cpu_inputMat);                                                      


	   int  multiWin    = 2;

	   int winSize      = cpu_config->windowHW;

	   int  stepSize    = cpu_config->step;

   
	    computeDisplacement_cuda(cpu_inputMat, multiWin, winSize, stepSize, cpu_disMat);            
	

		lowpassFilt_799_cuda (cpu_disMat);                                                    


		strainCalculate_cuda(cpu_disMat,   cpu_fitMat);                                     


		random_proess_cuda(cpu_fitMat, cpu_config, output);                                




		int   ss = 0;




}



