#include "hip/hip_runtime.h"
#include  "cuda_main.cuh"


#ifdef  _CUDA_MAIN_CUH
#include  "SysConfig.h"
#include  "CElasto.h"
#include  "FileUtility.h"
#include <fstream>
#include <string>
#include <iostream>
#include <time.h>
#include <>
#include <hip/device_functions.h>
#include <math_functions.h>
#include  <math.h>
#include <string.h>
#include <cstdio>
#include "opencv/highgui.h"
#include "opencv/cv.h"
#include "ImageFunc.h"


#endif


//�ں˺�����device����


//��Ҫ���ģ��ÿ�GPUоƬֻ��֧��1024��threads per  block !!!      changed  by  wong   2016/06/08

//�����������˲���    ���˲�           changed  by  wong    2016/5/13

__global__ void Bandpass_front_1(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)
{
	//float muData[40];

	//k  = blockIdx.x   i = blockIdx.y;
	float data_sum;

	float data_1;

	data_sum = 0.0;

	//	__shared__     float     data_sum[8192];



	/*for (int i = 0; i < iParaLen; i++)
	{
	muData[i] = *(param + i);
	}*/

	if (threadIdx.x <= iParaLen - 1)                                         //������ĿС�ڵ��ڳ�ͷ��Ŀ
	{

		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);      //b(0)*x(n-0)+b(1)*x(n-1)+...+b(n)*x(0)   

			data_sum += (data_1*param[i]);

		}

		data_1 = *(tInput + blockIdx.x * iWidth);                          // x(0)


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{
			data_sum += (data_1*param[j]);                                 //b(n+1)*x(0)+...+b(nb-2)*x(0)
		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x) = data_sum;



	}
	else                                                                  //������Ŀ���ڳ�ͷ��Ŀ            
	{
		//data_1 = (tInput + blockIdx.x*iWidth + blockIdx.y - threadIdx.x)->x;
		for (int i = 0; i <= iParaLen - 1; i++)
		{

			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);   //b(0)*x(n-0)+b(1)*x(n-1)+...+b(nb-2)*x(n-(nb-2))  

			data_sum += (data_1*param[i]);

		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x)= data_sum;

	}

}




//worker for  zero-phase filter  ,1024  threads  limited   .   changed   by  wong   2016/06/12

//test  result  : ok        wong    2016/06/13  

__global__ void Bandpass_front_1024(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)     {

	 float data_sum;

	 float data_1;

	 data_sum = 0.0;

	 int   line_serial;

	 int    bid = blockIdx.x;


	   line_serial = bid / 8;



	 int  line_mod = bid % 8;


//	 line_serial  = bid / 16;                            //changed   by wong 


//	 int  line_mod = bid % 16;                         //changed  by  wong 




	 if ((0 == line_mod))    {                                                                     //������                  

		 if ((threadIdx.x <= iParaLen - 1))                                                       //������ĿС�ڵ��ڳ�ͷ��Ŀ,�����ǳ�����
		 {

			 for (int i = 0; i <= threadIdx.x; i++)
			 {


				 data_1 = *(tInput + line_serial * iWidth + threadIdx.x - i);                  //b(0)*x(n-0)+b(1)*x(n-1)+...+b(n)*x(0)   

				 data_sum += (data_1*param[i]);

			 }


			 data_1 = *(tInput + line_serial * iWidth);                                      // x(0)


			 for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
			 {
				 data_sum += (data_1*param[j]);                                                    //b(n+1)*x(0)+...+b(nb-2)*x(0)
			 }

			 *(tOutPut + line_serial * iWidth + threadIdx.x) = data_sum;



		 }

		 else  if ((threadIdx.x > iParaLen - 1))   {                                               //������Ŀ���ڵ��ڳ�ͷ��Ŀ,�����ǳ�����

			 //data_1 = (tInput + blockIdx.x*iWidth + blockIdx.y - threadIdx.x)->x;
			 for (int i = 0; i <= iParaLen - 1; i++)
			 {

				 data_1 = *(tInput + line_serial *iWidth + threadIdx.x - i);                 //b(0)*x(n-0)+b(1)*x(n-1)+...+b(nb-2)*x(n-(nb-2))  

				 data_sum += (data_1*param[i]);

			 }

			 *(tOutPut + line_serial * iWidth + threadIdx.x) = data_sum;


		 }


	 }

	else                                                                                 //�ǳ����� (Ĭ��������Ŀ���ڵ��ڳ�ͷ��Ŀ)        
	{

		for (int i = 0; i <= iParaLen - 1; i++)
		{

			data_1    = *(tInput + blockIdx.x*blockDim.x+ threadIdx.x - i);   //b(0)*x(n-0)+b(1)*x(n-1)+...+b(nb-2)*x(n-(nb-2))  

			data_sum += (data_1*param[i]);

		}

		*(tOutPut + blockIdx.x*blockDim.x + threadIdx.x) = data_sum;

	}

//	 __syncthreads();
 


}




//��Ҫ���ģ��ÿ�GPUоƬֻ��֧��1024��threads per  block !!!      changed  by  wong   2016/06/08

//�����������˲���   ���˲�                          changed  by  wong     2016/5/13
__global__ void Bandpass_back_1(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)
{
	/*
	if (threadIdx.x < iParaLen - 1)                                     //  n  >= nb-1
	{
	return;                                                            //  �˴�������     changed  by  wong
	}
	//   changed   by  wong     2016/5/11
	*/                                                                 // �˴�û�п���С��nb-1���������Ҫ�������


	float data_1;

	float data_sum;

	data_sum = 0.0;



	if (threadIdx.x <= iParaLen - 1)   {                                //  ���ݳ���С�ڵ����˲�����ͷ����


		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);      //

			data_sum += (data_1*param[i]);

		}


		data_1 = *(tInput + blockIdx.x * iWidth + iWidth - 1);                        //  x(N-1) 


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{

			data_sum += (data_1*param[j]);                                            // b(n+1)*x(N-1)+...+b(nb-1)*x(N-1) 

		}


		*(tOutPut + blockIdx.x * iWidth + iWidth - 1 - threadIdx.x) = data_sum;                 // y(n)

	}

	else    {                                                                       //���ݳ��ȴ����˲�����ͷ����         


		for (int i = 0; i <= iParaLen - 1; i++)
		{
			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);

			data_sum += (data_1*param[i]);                                         //  y(N-1-n) = b(0)*x(N-1-n+0) +b(1)*x(N-1-n+1)+...+b(nb-1)*x(N-1-n+nb-1)

		}

		*(tOutPut + blockIdx.x * iWidth + iWidth - 1 - threadIdx.x)   = data_sum;




	}



	__syncthreads();  

}




//  worker for  zero-phase filter  ,1024  threads  limited   .   changed   by  wong   2016/06/12

//  test  result  : ok        wong    2016/06/13   

__global__ void Bandpass_back_1024(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)   {


	float  data_1;

	float  data_sum;

	data_sum = 0.0;


	int   line_serial;

	int    bid = blockIdx.x;


	line_serial = bid / 8;



	int  line_mod = bid % 8;


//	line_serial = bid / 16;


//	int  line_mod = bid % 16;







	if ((0 == line_mod))    {                                                                     // ������   


		if (threadIdx.x <= iParaLen - 1)   {                                                      // ���ݳ���С�ڵ����˲�����ͷ����,�����ǳ�����


			for (int i = 0; i <= threadIdx.x; i++)
			{

				 
				data_1 = *(tInput + line_serial*iWidth + iWidth - 1 - threadIdx.x + i);      // ��������

				data_sum += (data_1*param[i]);

			}


			data_1 = *(tInput + line_serial * iWidth + iWidth - 1);                          // x(N-1) 


			for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
			{

				data_sum += (data_1*param[j]);                                                   // b(n+1)*x(N-1)+...+b(nb-1)*x(N-1) 

			}


			*(tOutPut + line_serial * iWidth + iWidth-1 - threadIdx.x) = data_sum;              // y(n)      

		}

		else  if (threadIdx.x  >iParaLen - 1)   {                                              // ���ݳ��ȴ����˲�����ͷ����,�����ǳ�����         

			data_sum = 0;

			for (int i = 0; i <= iParaLen - 1; i++)
			{
				data_1 = *(tInput + line_serial*iWidth + iWidth - 1 - threadIdx.x + i);

				data_sum += (data_1*param[i]);                                               //  y(N-1-n) = b(0)*x(N-1-n+0) +b(1)*x(N-1-n+1)+...+b(nb-1)*x(N-1-n+nb-1)

			}

			*(tOutPut + line_serial * iWidth + iWidth - 1 - threadIdx.x) = data_sum;




		}



	}  

	else  {                                                                                 //  �ǳ����� (Ĭ��������Ŀ���ڵ��ڳ�ͷ��Ŀ)    

		    data_sum = 0;
		  

		for (int i = 0; i <= iParaLen - 1; i++)
		{
			data_1 = *(tInput + line_serial*iWidth + iWidth - 1 - (threadIdx.x + line_mod*blockDim.x) + i);

			data_sum += (data_1*param[i]);                                               //  y(N-1-n) = b(0)*x(N-1-n+0) +b(1)*x(N-1-n+1)+...+b(nb-1)*x(N-1-n+nb-1)

		}

		*(tOutPut + line_serial * iWidth + iWidth - 1 - (threadIdx.x + line_mod*blockDim.x)) = data_sum;




	}
	
	
	
	
	}





















//changed   by  wong      2016/5/13

__device__      void   xcorr_cuda(const  Complex* templateMat_startID, const Complex* objectMat_startID, Complex*resultMat_startID)     {


	for (int i = 0; i < 101; i++)   {


		Complex     sum_object = 0;

		Complex     frac_object = 0;


		Complex     pow_template = 0;


		Complex      pow_object = 0;


		Complex     result = 0;


		//sum_object 

		for (int j = 0; j < 100; j++)  {


			sum_object += *(objectMat_startID + i + j);


		}

		//  ave

		Complex   ave_object =   sum_object / 100;


		//fraction

		for (int j = 0; j < 100; j++)  {

			Complex    tmp = *(templateMat_startID + j) *  (*(objectMat_startID + i + j) - ave_object);


			frac_object += tmp;

		}


		//pow   temp

		for (int j = 0; j < 100; j++)  {


			pow_template += *(templateMat_startID + j) * *(templateMat_startID + j);

		}

		//pow   objectMat 

		for (int j = 0; j < 100; j++)  {


			pow_object += *(objectMat_startID + i + j)* * (objectMat_startID + i + j);

		}

		//result

		result = sqrt(pow_template*pow_object);

		//output

		*(resultMat_startID + i) = frac_object / result;

	}


}


//changed   by  wong    2016/5/13

__device__      void   minMax_cuda(Complex*resultMat_startID, Complex* min_value, Complex*  max_value, int * max_location)   {

//	int      max_loc_temp = 0;

//	int      min_loc_temp = 0;

//	float      max_temp   = 0;

//	float      min_temp   = 0;

	//�����ֵ��λ��

	for (int i = 0; i < 101; i++)  {

		if (*(resultMat_startID + i) >= *max_value)  {

			*max_location   = i;

			*max_value = *(resultMat_startID + i);



		}

	}

	//����Сֵ��λ��

//	for (int i = 0; i < 101; i++)  {

//		if (*(resultMat_startID + i) <= *min_value)  {

		//	min_loc_temp = i;

//			*min_value = *(resultMat_startID + i);

			
//		}


//	}

	//���

//	*min_value = min_temp;

//	*max_value = max_temp;

//	max_location = max_loc_temp;

}


 
//changed  by   wong      2016/5/17

__device__    void    interp_cuda(Complex*resultMat_startID, int *  max_loc, Complex*max_value, int * multiWin, int * winSize, Complex*  displace)     {

	Complex*pre = (Complex*)resultMat_startID + *max_loc - 1;

	Complex*next = (Complex*)resultMat_startID + *max_loc + 1;


	*displace   = (*multiWin - 1) * *winSize / 2 - *max_loc - (*pre - *next) / (2 * (*pre - 2 * *max_value + *next));


}





// ���Ϊλ��299*799����

// test   result :  ok     wong   2016/06/24

__global__   void  displacement_api_cuda(Complex*disInputCuda, int rows, int cols, int  multiWin, int winSize, int  stepSize, templateMat*templateMatShare, objectMat* objectMatShare, resultMat*resultMatShare, Complex*min, Complex*max, int*max_location, Complex* displacement )      {


	int   out_offset = blockIdx.x *blockDim.x + threadIdx.x;                     // ���λ�ƾ���ƫ��ֵ

	int    bid       = blockIdx.x ;                                              //  ��Ӧblock ID 
	
	int    tid       = threadIdx.x;                                             //   ��Ӧthread ID 


   //����ʹ��3D���飬��Ϊ�����ڴ治���ã�ֻ��49152���ֽ�each block 

	//�����ڴ�

	//����ȫ���ڴ棡����

//	__shared__     Complex*     templateMatShare[THREAD_NUM];        //100�׵�ַ

//	__shared__     Complex*     objectMatShare[THREAD_NUM];          //200�׵�ַ

//	__shared__     Complex*     resultMatShare[THREAD_NUM];          //101�׵�ַ 


//	__shared__     templateMat   templateMatShare[THREAD_NUM];

//	__shared__     objectMat     objectMatShare[THREAD_NUM];

//	__shared__     resultMat     resultMatShare[THREAD_NUM];


//	  Complex*templateMatShare;                          //   ģ���ڴ���GPU����         ���Ǿֲ�����                  


//	  Complex*objectMatShare;                           //    Ŀ���ڴ���GPU����         ���Ǿֲ�����


//	  Complex*resultMatShare;                           //    ƥ������GPU����         ���Ǿֲ�����




//	hipMalloc(&templateMatShare, winSize* sizeof(Complex));             //ģ�����


//	hipMalloc(&objectMatShare, winSize*multiWin* sizeof(Complex));     //Ŀ�����


//	hipMalloc(&resultMatShare, (winSize + 1)* sizeof(Complex));        //�������




//	 templateMatShare[out_offset].elem   = (Complex*)(disInputCuda + blockIdx.x*cols + (multiWin - 1) * winSize / 2 + threadIdx.x * stepSize);
		







	    Complex*templateMatID;                               //ID


	  Complex*objectMatID;                                //ID



	//12784�ֽ�

//	__shared__     Complex*     min[THREAD_NUM];

//	__shared__     Complex*     max[THREAD_NUM];

//	__shared__      int        max_location[THREAD_NUM];

//	__shared__    Complex*     displacement[THREAD_NUM];




	//׼���������      �߳̿������� �����ù����ڴ�


	//	(templateMat*)(templateMat_startID + threadIdx.x)->elem = (Complex*)(disInputCuda + blockIdx.x*cols + (multiWin - 1) * winSize / 2 + threadIdx.x * stepSize);

	      templateMatID = (Complex*)(disInputCuda + blockIdx.x*cols + (multiWin - 1) * winSize / 2 + threadIdx.x * stepSize);




		  for (int i = 0; i < 100;i++)  {

			  if (i < 64)    {
			  
				  templateMatShare[out_offset].tempData.elem[i]= *(templateMatID + i);
			  
			  }
		  
			  else
				      

			    templateMatShare[out_offset].tempData.atom[i-64] = *(templateMatID + i);
		  
		  
		  }



		  /*      change   by wong 

		  for (int i = 0; i < 64; i++) {

		   templateMatShare[out_offset].elem[i] = *(templateMatID+i);


	//		 *(templateMatShare[out_offset].elem+i)  = *(templateMatID + i);

		  }


		  for (int j = 0; j < 36; j++) {

			  templateMatShare[out_offset].atom[j] = *(templateMatID + j+64);


			  //		 *(templateMatShare[out_offset].elem+i)  = *(templateMatID + i);

		  }      change  by  wong 

    */       




		  objectMatID   = (Complex*)(disInputCuda + (blockIdx.x + 1)*cols + threadIdx.x * stepSize);


   
		  for (int i = 0; i < 200; i++)  {

			  if (i<64)
				  objectMatShare[out_offset].objData.elem_0[i]     = *(objectMatID + i);
			  else if (i<128)
				  objectMatShare[out_offset].objData.elem_1[i - 64] = *(objectMatID + i);

			  else if (i<192)
				  objectMatShare[out_offset].objData.elem_2[i - 128] = *(objectMatID + i);
			  else
				  objectMatShare[out_offset].objData.atom[i - 192]   = *(objectMatID + i);

			  //	  *(objectMatShare[out_offset].elem + i) = *(objectMatID + i);

		  }

       



		  for (int i = 0; i < 101; i++)  {
		   
			  if (i<64)
				  resultMatShare[out_offset].resData.elem[i]   = 0;
		   
			  else
				  resultMatShare[out_offset].resData.atom[i-64] = 0;
		  
		  }
		 





		  /*    change  by wong  
             
		  for (int i = 0; i < 192; i++)  {
		  
			  if (i<64) 
			  objectMatShare[out_offset].elem_0[i]       = *(objectMatID+i);
			  else if (i<128)
			  objectMatShare[out_offset].elem_1[i-64]    = *(objectMatID + i);

			  else 
			  objectMatShare[out_offset].elem_2[i - 128] = *(objectMatID + i);

		//	  *(objectMatShare[out_offset].elem + i) = *(objectMatID + i);
		  
		  }
  


		  for (int j = 0; j < 8; j++) {

			  objectMatShare[out_offset].atom[j] = *(templateMatID + j + 192);


			  //		 *(templateMatShare[out_offset].elem+i)  = *(templateMatID + i);

		  }      change  by  wong 
		   

   */
   




//	__syncthreads();

//	hipDeviceSynchronize();

//	hipDeviceSynchronize();



	//�������

		  xcorr_cuda(templateMatShare[out_offset].tempData.elem, objectMatShare[out_offset].objData.elem_0, resultMatShare[out_offset].resData.elem);


	//	__syncthreads();

//	hipDeviceSynchronize();


	//�������ֵ

		minMax_cuda(resultMatShare[out_offset].resData.elem, &min[out_offset], &max[out_offset], &max_location[out_offset]);


//	__syncthreads();

//	hipDeviceSynchronize();

	//��ֵ

		interp_cuda(resultMatShare[out_offset].resData.elem, &max_location[out_offset], &max[out_offset], &multiWin, &winSize, &displacement[out_offset]);


//		__syncthreads();

//	hipDeviceSynchronize();


	//ȥ����


	//λ�Ƶ���


	//��ֵ�˲�


	//�����ֵ

	//  *��disOutputCuda+bid��     =    displacement   ��

//	disOutputCuda[out_offset] = *displacement[threadIdx.x];


}



//ȥ����        changed  by wong    2016/5/18

__global__  void   remove_singular_cuda(Complex*disOutputCuda, Complex*singularOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                                           // ���λ�ƾ���ƫ��ֵ

	int    bid   = blockIdx.x;                                                                     //  block   id

	int    tid   = threadIdx.x;                                                                    //  thread  id    

//	int   offrow =( bid  > 0 ) ? (blockIdx.x - 1)*blockDim.x + threadIdx.x  : 0;                   // ��һ�����λ�ƾ���ƫ��ֵ  �˴����޸�  wong    2016/06/24

	int    offrow = 0;

	if (bid  > 0 && bid < gridDim.x - 1 && tid < blockDim.x-1 )   {
	
		    offrow = (blockIdx.x - 1)*blockDim.x + threadIdx.x;
	
	} 




	if (bid > 0 && bid < gridDim.x - 1 && tid < blockDim.x - 1 && (abs(disOutputCuda[offset]) > 12))  {

		singularOutputCuda[offset] = disOutputCuda[offrow];

	}

	else  {

		singularOutputCuda[offset] = disOutputCuda[offset];

	}


}


//λ�Ƶ���       changed   by  wong    2016/5/18

__global__   void   displace_add_cuda(Complex*singularOutputCuda, Complex*addOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                               // ���λ�ƾ���ƫ��ֵ

	int    bid   = blockIdx.x;                                                          // block   id

	int    tid   = threadIdx.x;                                                         // thread  id   

	int   offrow = (bid >0 ) ? ( (blockIdx.x - 1)*blockDim.x + threadIdx.x)  :0 ;       // ��һ�����λ�ƾ���ƫ��ֵ

	int   nextoff =  (blockIdx.x + 1)*blockDim.x + threadIdx.x;


	Complex  sum = 0.0;




	if (bid > 0)  {

		       //new  changed  

		for (int i = 0; i < bid; i++)   {

			int  off = i*blockDim.x + threadIdx.x;


			sum = sum + singularOutputCuda[off];

		}


		addOutputCuda[offset] = singularOutputCuda[offset] + sum;


	}

	else   {

		addOutputCuda[offset] = singularOutputCuda[offset];

	}







//	if (bid < gridDim.x - 1)     {
	
	    
//		addOutputCuda[nextoff] = singularOutputCuda[nextoff] + singularOutputCuda[offset];
	
	
//	}








}



//������չN-1�У���������

__global__   void   extend_data_cuda(Complex*addOutputCuda, Complex*extendOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                        // ���λ�ƾ���ƫ��ֵ

	int    bid   = blockIdx.x;                                                  // block   id

	int    tid   = threadIdx.x;                                                 // thread  id   


	if (tid<N - 1)  {

		int   add_base = blockIdx.x *(blockDim.x - (N - 1));

		extendOutputCuda[offset] = addOutputCuda[add_base];                    //  extend  primites

	}

	else
	{

		int   extoff = blockIdx.x *(blockDim.x - (N - 1)) + threadIdx.x - (N - 1);

		extendOutputCuda[offset] = addOutputCuda[extoff];

	}

}


//���ۼ�ƽ��   

__global__ void  smooth_filter_cuda(Complex*extendOutputCuda, Complex* smoothOutputCuda)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                        // ���λ�ƾ���ƫ��ֵ

	int   extbase = blockIdx.x*(blockDim.x + N - 1) + threadIdx.x;              // ��ַ

	int    bid = blockIdx.x;                                                    // block   id

	int    tid = threadIdx.x;                                                   // thread  id  


	Complex   sum = 0;


	for (int i = extbase; i < extbase + N; i++)  {


		Complex  temp = *(extendOutputCuda + i);

		sum = sum + temp;


	}

	smoothOutputCuda[offset] = sum / N;

}




__global__  void   timeField_filter_cuda(const Complex* smoothOutputCuda, const float* param,  const int  steps, Complex* timeFilterOutputCuda)    {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                        // ���λ�ƾ���ƫ��ֵ


	int    bid = blockIdx.x;                                                    // block   id

	int    tid = threadIdx.x;                                                   // thread  id     

	Complex  sum_temp = 0;

	float    coeff   = 0;

	for (int i = 0; i <= bid; i++)   {

		if ((bid - i) < steps)

			coeff = param[bid - i];

		else

			coeff = param[0];


		sum_temp += smoothOutputCuda[i*blockDim.x + threadIdx.x] * coeff;


	}

	timeFilterOutputCuda[offset] = sum_temp;

}






bool    CudaMain::isAvailable()  {

	int   count = 0;

	printf("Start to detecte devices.........\n");                   //  ��ʾ��⵽���豸��

	hipGetDeviceCount(&count);                                     //   �������������ڵ���1.0���豸��

	if (count == 0){

		fprintf(stderr, "There is no device.\n");

		return false;

	}


	printf("%d device/s detected.\n", count);                      //   ��ʾ��⵽���豸��


	int i;

	for (i = 0; i < count; i++){                                  //  ������֤��⵽���豸�Ƿ�֧��CUDA

		hipDeviceProp_t prop;

		if (hipGetDeviceProperties(&prop, i) == hipSuccess) {  //  ����豸���Բ���֤�Ƿ���ȷ

			if (prop.major >= 1)                                 //  ��֤�����������������������ĵ�һλ���Ƿ����1

			{
				printf("Device %d: %s supports CUDA %d.%d.\n", i + 1, prop.name, prop.major, prop.minor);//��ʾ��⵽���豸֧�ֵ�CUDA�汾
				break;


			}
		}
	}

	if (i == count) {                                         //   û��֧��CUDA1.x���豸
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);                                       //    �����豸Ϊ�����̵߳ĵ�ǰ�豸

	return true;

}





CudaMain::CudaMain()  {

	


	cpu_inputMat      = NULL;
	
	cpu_SplineOutMat  = NULL ;

	cpu_RadonMat      = NULL;

	cpu_WaveRate      = 0  ;

	mallocFlag        = false;

	cpu_config        = new   ConfigParam ;

	cpu_disMat        = NULL;



//	memset(cpu_config, 0, sizeof(cpu_config));




	inputMat         = NULL;

	zeroFilterMat    = NULL;

	frontFilterMat   = NULL;

	disOutput        = NULL;

	bandfilterParam  = NULL;

	lowfilterParam   = NULL;

	matchfilterParam = NULL;

	lowFrontMat      = NULL;

	lowBackMat       = NULL;

	singularOutputCuda = NULL;

	addOutputCuda      = NULL;

	extendOutputCuda   = NULL;


	radonIn          = NULL;

	radonOut         = NULL;


}




CudaMain :: ~CudaMain()  {

	freeMem(); 



}



void   CudaMain::inputConfigParam( ConfigParam*config) {



	cpu_config = config;


}



void  CudaMain::inputRfData(const EInput& in) {     //�������ݵ�cpu_inputMat��

	float* input = in.pDatas;


	for (int i = 0; i < cpu_inputMat->rows; i++)
	{
		for (int j = 0; j < cpu_inputMat->cols; j++)
		{
			*(static_cast<float*>(static_cast<void*>(CV_MAT_ELEM_PTR(*cpu_inputMat, i, j)))) = input[i * cpu_inputMat->cols + j];
		}
	}


}




void  CudaMain::getbandFilterParam(std::string paramFileName) {

	if (paramFileName.size() == 0)
	{
		exit(1);
	}

	std::fstream paramFile(paramFileName.c_str());

	if (!paramFile)
	{
		exit(1);
	}

	float tmp;

	std::string str;


	cpu_bandfilterParam.clear();

	while (!paramFile.eof())
	{
		paramFile >> tmp;
		cpu_bandfilterParam.push_back(tmp);
	}
	paramFile.close();



}


void   CudaMain::getlowFilterParam(std::string paramFileName) {

	if (paramFileName.size() == 0)
	{
		exit(1);
	}

	std::fstream paramFile(paramFileName.c_str());

	if (!paramFile)
	{
		exit(1);
	}

	float tmp;

	std::string str;


	cpu_lowfilterParam.clear();

	while (!paramFile.eof())
	{
		paramFile >> tmp;
		cpu_lowfilterParam.push_back(tmp);
	}
	paramFile.close();


}



void  CudaMain::getmatchFilterParam(std::string paramFileName) {

	if (paramFileName.size() == 0)
	{
		exit(1);
	}

	std::fstream paramFile(paramFileName.c_str());

	if (!paramFile)
	{
		exit(1);
	}

	float tmp;

	std::string str;


	cpu_matchfilterParam.clear();

	while (!paramFile.eof())
	{
		paramFile >> tmp;
		cpu_matchfilterParam.push_back(tmp);
	}
	paramFile.close();

}











void  CudaMain::mallocMem(void)  {

	mallocMats();

	mallocGPUMem();

	

}



void CudaMain::freeMem(void)  {

	freeMats();
   
	deleteGPUMem();
}





void   CudaMain::mallocGPUMem() {



	int  MatRows = cpu_config->shearFrameLineNum;

	int  MatCols = cpu_config->sampleNumPerLine ;

	int windowHW = cpu_config->windowHW;

	int maxLag   = cpu_config->maxLag;

	int step     = cpu_config->step;


	int interpnum  = cpu_config->fitline_pts;

	int iBPParaLen = 40;                                                      // bandpassfilter�ĳ��ȣ�

	iBPParaLen     = (iBPParaLen > cpu_bandfilterParam.size()) ? iBPParaLen : cpu_bandfilterParam.size();


	int iLPParaLen = 40;                                                      // lowpassfilter�ĳ��ȣ�

	iLPParaLen     = (iBPParaLen > cpu_lowfilterParam.size()) ? iBPParaLen : cpu_lowfilterParam.size();


	int iMHParaLen = 40;                                                      // matchfilter�ĳ��ȣ�

	iMHParaLen    = (iBPParaLen > cpu_matchfilterParam.size()) ? iBPParaLen : cpu_matchfilterParam.size();






	if (MatRows == 0 || MatCols == 0)
	{

		printf("  row  and col  is zero! call InputConfigParas first!\n");
		return;

	}

	hipError_t cudaStatus = hipSetDevice(0);                                 // 0��titan�Կ�

	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		return;
	}



	int  multiWin = 2;                                                      //  �󴰿ڶ�С���ڵı���

	int cxorrLines = MatRows - 1;                                           //  λ�ƾ������ɨ������Ŀ        299

	int iOutRows = (MatCols - multiWin*windowHW) / step;                    //  λ�ƾ��������Ҫƥ��Ķ���     799 

	int extRows = iOutRows + N - 1;                                         //  ��չ����  799+100-1

	hipMalloc(&disOutput, cxorrLines *iOutRows* sizeof(Complex));          //  λ�ƾ���GPU�ڴ����


//	hipMalloc(&templateMatShare, cxorrLines *iOutRows* sizeof(templateMat));       //ģ�������GPUȫ���ڴ����


//	hipMalloc(&objectMatShare, cxorrLines *iOutRows* sizeof(objectMat));         //Ŀ�������GPUȫ���ڴ����


//	hipMalloc(&resultMatShare, cxorrLines *iOutRows* sizeof(resultMat));        //���������GPUȫ���ڴ����






	hipMalloc(&singularOutputCuda, cxorrLines *iOutRows* sizeof(Complex)); // ȥ���������GPU�ڴ����


	hipMalloc(&addOutputCuda, cxorrLines *iOutRows* sizeof(Complex));      // λ�Ƶ�����GPU�ڴ����


	hipMalloc(&extendOutputCuda, cxorrLines *extRows* sizeof(Complex));     // ��չ������GPU�ڴ����



	hipMalloc(&inputMat, MatRows * MatCols * sizeof(Complex));             //   ���������GPU�϶�Ӧ���ڴ棻


	hipMalloc(&zeroFilterMat, MatRows * MatCols * sizeof(Complex));       //   ��ͨ����λ�˲������GPU�ڴ���䣻


	hipMalloc(&frontFilterMat, MatRows * MatCols * sizeof(Complex));     //  ��ͨ����λ���˲���GPU�ڴ����



	hipMalloc(&lowBackMat, cxorrLines * iOutRows * sizeof(Complex));       //   ��ͨ����λ�˲������GPU�ڴ���䣻


	hipMalloc(&lowFrontMat, cxorrLines * iOutRows * sizeof(Complex));     //  ��ͨ����λ���˲���GPU�ڴ����

	


	hipMalloc(&bandfilterParam, iBPParaLen * sizeof(float));                // iBPParaLen�˲�������40


	hipMalloc(&lowfilterParam, iLPParaLen * sizeof(float));                // iLPParaLen�˲�������40


	hipMalloc(&matchfilterParam, iMHParaLen * sizeof(float));              // iMHParaLen�˲�������40



	hipMalloc(&fit_IN, cxorrLines *iOutRows* sizeof(Complex));          //  λ�ƾ���GPU�ڴ����


	int   points = 5;


	int   strain_col = iOutRows - points + 1;

	hipMalloc(&fit_Out, cxorrLines *strain_col* sizeof(Complex));          //  λ�ƾ���GPU�ڴ����






	int RadonInputCols      = 1961;                                     // 1961

	int RadonInputRows      = 4;                                       // 4

	hipMalloc(&radonIn, sizeof(float) * RadonInputCols * RadonInputRows);                //�����任GPU����

	hipMalloc(&radonOut, sizeof(float) * RadonInputCols * (RadonInputCols - 1));        //�����任GPU���  


	mallocFlag             = true;




}





void  CudaMain::deleteGPUMem()  {


	if (inputMat != NULL)
	{
		hipFree(inputMat);

		inputMat = NULL;
	}

	
	if (zeroFilterMat != NULL)
	{
		hipFree(zeroFilterMat);
		zeroFilterMat = NULL;
	}


	if (frontFilterMat != NULL)
	{
		hipFree(frontFilterMat);
		frontFilterMat = NULL;
	}


	if (lowBackMat != NULL)
	{
		hipFree(lowBackMat);
		lowBackMat = NULL;
	}


	if (lowFrontMat != NULL)
	{
		hipFree(lowFrontMat);
		lowFrontMat = NULL;
	}




	if (disOutput != NULL)
	{
		hipFree(disOutput);
		disOutput = NULL;
	}




	if (singularOutputCuda != NULL)
	{
		hipFree(singularOutputCuda);

		singularOutputCuda = NULL;
	}


	if (addOutputCuda != NULL)
	{
		hipFree(addOutputCuda);

		addOutputCuda = NULL;
	}


	if (extendOutputCuda != NULL)
	{
		hipFree(extendOutputCuda);

		extendOutputCuda = NULL;
	}







	if (bandfilterParam != NULL)
	{
		hipFree(bandfilterParam);
		bandfilterParam = NULL;
	}

	
	if (lowfilterParam != NULL)
	{
		hipFree(lowfilterParam);
		lowfilterParam = NULL;
	}



	if (matchfilterParam != NULL)
	{
		hipFree(matchfilterParam);
		matchfilterParam = NULL;
	}






	if (radonIn != NULL)
	{
		hipFree(radonIn);
	}

	if (radonIn != NULL)
	{
		hipFree(radonIn);
	}


	hipDeviceReset();

	mallocFlag = false;


}




void  CudaMain::mallocMats() {


	cpu_inputMat    =   cvCreateMat(cpu_config->shearFrameLineNum, cpu_config->sampleNumPerLine, CV_32FC1);         //�������

	int  MatRows    = cpu_config->shearFrameLineNum;

	int  MatCols    = cpu_config->sampleNumPerLine;

	int windowHW    = cpu_config->windowHW;

	int maxLag      = cpu_config->maxLag;

	int step        = cpu_config->step;


	int  multiWin   = 2;                                                    //  �󴰿ڶ�С���ڵı���

	int cxorrLines  = MatRows - 1;                                         //   λ�ƾ������ɨ������Ŀ        299

	int iOutRows    = (MatCols - multiWin*windowHW) / step;               //    λ�ƾ��������Ҫƥ��Ķ���     799 

	cpu_disMat      = cvCreateMat(cxorrLines, iOutRows, CV_32FC1);       //     λ�ƾ���   



	int  fit_point  = 5;

	
	int  fit_cols = iOutRows - fit_point + 1;

	cpu_fitMat = cvCreateMat(cxorrLines, fit_cols, CV_32FC1);                 





	cpu_SplineOutMat = cvCreateMat(1962, 4, CV_32FC1);                  //    SplineOutMat��������ڻ�ͼ���ȽϽ��  

		
	cpu_RadonMat    = cvCreateMat(1962, 4, CV_32FC1);                  //     radon������Ƚϼ�����  



	mallocFlag     = false; 
	

//	cpu_config     = (ConfigParam*)malloc(1 * sizeof(ConfigParam));     

	
//	memset(cpu_config, 0, sizeof(cpu_config));

}



void   CudaMain::freeMats() {

	if (cpu_inputMat != NULL)
	{
		cvReleaseMat(&cpu_inputMat);
		cpu_inputMat = NULL;
	}
	

	if (cpu_disMat != NULL)
	{
		cvReleaseMat(&cpu_disMat);
		cpu_disMat = NULL;
	}
	

	if (cpu_SplineOutMat != NULL)
	{
		cvReleaseMat(&cpu_SplineOutMat);
		cpu_SplineOutMat = NULL;
	}

	if (cpu_RadonMat != NULL)
	{
		cvReleaseMat(&cpu_RadonMat);
		cpu_RadonMat = NULL;
	}
	

	mallocFlag = NULL;


	free(cpu_config);


	cpu_config = NULL;

	  
}








//this   threads  number  of  this   module    is  8192  .not  suitable  for   this   GTX560 TI  GPU  platform  

CvMat*  CudaMain::bandpassFilt_cuda(CvMat* rawMat)  {


	Complex* h_MatData = (Complex*)rawMat->data.fl;

	hipMemsetAsync(frontFilterMat, 0, sizeof(Complex)*rawMat->cols*rawMat->rows);

	hipMemcpyAsync(zeroFilterMat, h_MatData, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyHostToDevice);    //����CPU��RF���ݵ�GPU

	int steps = cpu_bandfilterParam.size();

	hipMemcpyAsync(bandfilterParam, &cpu_bandfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);                  //����CPU�г�ͷ���ݵ�GPU 





	dim3 blockID, threadID;

	blockID.x  = rawMat->rows;

	threadID.x = rawMat->cols;

	hipDeviceSynchronize();

	Bandpass_front_1 <<<blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);

	hipDeviceSynchronize();


	hipMemcpy(zeroFilterMat, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToDevice);


	Bandpass_back_1 << <blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);


	hipDeviceSynchronize();

	   
	hipFree(bandfilterParam);

	hipMemcpy(h_MatData, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

	hipFree(zeroFilterMat);

	hipFree(frontFilterMat);


	SaveDataFile("bpfilt.dat", rawMat);


	return rawMat;


}



//worker  for  this   platform   ,1024   threads   limited .  change   by  wong     2016/6/12

CvMat*  CudaMain::bandpassFilt_1024_cuda(CvMat* rawMat)  {


	Complex* h_MatData = (Complex*)rawMat->data.fl;

	hipMemsetAsync(frontFilterMat, 0, sizeof(Complex)*rawMat->cols*rawMat->rows);

	hipMemcpyAsync(zeroFilterMat, h_MatData, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyHostToDevice);    //����CPU��RF���ݵ�GPU

	int steps = cpu_bandfilterParam.size();

	hipMemcpyAsync(bandfilterParam, &cpu_bandfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);                  //����CPU�г�ͷ���ݵ�GPU 


	dim3 blockID, threadID;

	blockID.x = rawMat->rows*8;                           //changed   by  wong  


//	blockID.x = rawMat->rows * 8*2;


	threadID.x = rawMat->cols/8;                      //changed  by  wong 

//	threadID.x = rawMat->cols / 16;





	hipDeviceSynchronize();

	Bandpass_front_1024 << <blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);

	hipDeviceSynchronize();


	//test  for  line 2 

//	hipMemcpy(h_MatData, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

//	SaveDataFile("front_1024.dat", rawMat);


	hipMemcpy(zeroFilterMat, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToDevice);


	Bandpass_back_1024 << <blockID, threadID >> >(zeroFilterMat, rawMat->cols, bandfilterParam, steps, frontFilterMat);



	hipDeviceSynchronize();


	hipFree(bandfilterParam);

	hipMemcpy(h_MatData, frontFilterMat, sizeof(Complex)*rawMat->cols*rawMat->rows, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

	hipFree(zeroFilterMat);

	hipFree(frontFilterMat);


	//SaveDataFile("back_1024.dat", rawMat);


	return rawMat;
 






}










void  CudaMain::zeroFilter_cuda(CvMat* rawMat, Complex*filterOutput) {







}





void   CudaMain::computeDisplacement_cuda(CvMat* filtOutMat, int  multiWin, int winSize, int stepSize, CvMat*outputMat){

//	CvMat*outputMat = 0;

	int     WinNum    = (filtOutMat->cols - multiWin*winSize) / stepSize;       //  һάλ�ƾ���

	Complex* hInput   = (Complex*)filtOutMat->data.fl;                         //   ����λ�ã�

	Complex*hOutput  = (Complex*)outputMat->data.fl;                        //   ����λ�ã�


	hipMemcpy(inputMat, hInput, filtOutMat->cols*filtOutMat->rows*sizeof(Complex), hipMemcpyHostToDevice);   //  CPU-GPU

	dim3 dBlock;

	dim3 dThread;

	dBlock.x = filtOutMat->rows - 1;                                 // ����������� ,����        299

//	dBlock.x = 200;                                                 //just   test   wong    2016/06/15

	dThread.x = WinNum;                                             // ����������� , �߳���      799


//	__device__   Complex*templateMatShare;                          //   ģ���ڴ���GPU����         ���Ǿֲ�����                  


//	__device__   Complex*objectMatShare;                           //    Ŀ���ڴ���GPU����         ���Ǿֲ�����


//	__device__   Complex*resultMatShare;                           //    ƥ������GPU����         ���Ǿֲ�����




	templateMat*templateMatShare;                                 //   ģ���ڴ���GPU���� 


	objectMat* objectMatShare;                                   //    Ŀ���ڴ���GPU���� 



	resultMat*resultMatShare;                                   //    ƥ������GPU���� 



	Complex*      min;


	Complex*      max;

	int*          max_location;


	Complex*      displacement;








	hipMalloc(&templateMatShare, dBlock.x*dThread.x* sizeof(templateMat));             //ģ�������GPUȫ���ڴ����


	hipMalloc(&objectMatShare,  dBlock.x*dThread.x* sizeof(objectMat));               //Ŀ�������GPUȫ���ڴ����


	hipMalloc(&resultMatShare,  dBlock.x*dThread.x* sizeof(resultMat));             //���������GPUȫ���ڴ����



	hipMalloc(&min, dBlock.x*dThread.x* sizeof(Complex));                           // min��GPUȫ���ڴ����


	hipMalloc(&max, dBlock.x*dThread.x* sizeof(Complex));                          // max��GPUȫ���ڴ����


	hipMalloc(&max_location, dBlock.x*dThread.x* sizeof(int));                     // max_location��GPUȫ���ڴ����


	hipMalloc(&displacement, dBlock.x*dThread.x* sizeof(Complex));                // max_location��GPUȫ���ڴ����



	
	//��λ�ƾ���  

	displacement_api_cuda << < dBlock, dThread >> >   (inputMat, filtOutMat->rows, filtOutMat->cols, multiWin, winSize, stepSize, templateMatShare, objectMatShare, resultMatShare, min, max, max_location, displacement);

	hipDeviceSynchronize();

	//test  for   displace      changed  by  wong   2016/06/20

	hipMemcpy(hOutput, displacement, sizeof(Complex)*outputMat->cols*outputMat->rows, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

	SaveDataFile("weiyi_gpu.dat", outputMat);




	hipFree(templateMatShare);

	hipFree(objectMatShare);

	hipFree(resultMatShare);

	hipFree(min);

	hipFree(max);

	hipFree(max_location);









	//ȥ����                                   

	remove_singular_cuda << <dBlock, dThread >> >   (displacement, singularOutputCuda);

	hipDeviceSynchronize();


	//test  for   displace      changed  by  wong   2016/06/20

//	hipMemcpy(hOutput, singularOutputCuda, sizeof(Complex)*outputMat->rows*outputMat->cols, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

//	 SaveDataFile("sigular_gpu.dat", outputMat);





	//λ�Ƶ���                 

	displace_add_cuda << <dBlock, dThread >> >  (singularOutputCuda, addOutputCuda);

	hipDeviceSynchronize();


	//test  for   add      changed  by  wong   2016/06/20

//	hipMemcpy(hOutput, addOutputCuda, sizeof(Complex)*outputMat->rows*outputMat->cols, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

//    SaveDataFile("add_gpu.dat", outputMat);






	//ǰN-1�в�0    

	int  ext_threads = dThread.x + N - 1;

	extend_data_cuda << < dBlock, ext_threads >> > (addOutputCuda, extendOutputCuda);

	hipDeviceSynchronize();

	hipFree(addOutputCuda);

	//ƽ���˲�  

	smooth_filter_cuda << <dBlock, dThread >> >   (extendOutputCuda, disOutput);

	hipDeviceSynchronize();

	hipFree(extendOutputCuda);


	//test  for   smmoth      changed  by  wong   2016/06/20

//	hipMemcpy(hOutput, disOutput, sizeof(Complex)*outputMat->rows*outputMat->cols, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

///	SaveDataFile("smooth_gpu.dat", outputMat);








	//ʱ���˲���ƥ���˲���50Hz��ǿ     ���� param, iParaLen, steps ʹ�ó����ڴ� 

	int steps = cpu_matchfilterParam.size();

	hipMemcpyAsync(matchfilterParam, &cpu_matchfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);             //����CPU�г�ͷ���ݵ�GPU 


	timeField_filter_cuda << <dBlock, dThread >> > (disOutput, matchfilterParam,  steps, singularOutputCuda);

	hipDeviceSynchronize();

	hipFree(disOutput);

	//��GPU������CPU�ڴ�


	hipMemcpy(hOutput, singularOutputCuda, dBlock.x  * dThread.x*sizeof(Complex), hipMemcpyDeviceToHost);   //  GPU-CPU


	hipFree(singularOutputCuda);

	
	//test  for   smmoth      changed  by  wong   2016/06/20

    SaveDataFile("time_gpu.dat", outputMat);


}






void   CudaMain::zeroDisplacement_cuda(CvMat* inputMat, int  multiWin, int winSize, int stepSize, Complex*disOutput){





}


//changed   by  wong     2016/06/22         lowpass filter    size  : 299*799

//���˲�   ������

__global__ void     lowpass_front_799(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)    {


	float data_sum;

	float data_1;

	data_sum = 0.0;


	if (threadIdx.x <= iParaLen - 1)                                         //������ĿС�ڵ��ڳ�ͷ��Ŀ
	{

		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);      //b(0)*x(n-0)+b(1)*x(n-1)+...+b(n)*x(0)   

			data_sum += (data_1*param[i]);

		}

		data_1 = *(tInput + blockIdx.x * iWidth);                          // x(0)


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{
			data_sum += (data_1*param[j]);                                 //b(n+1)*x(0)+...+b(nb-2)*x(0)
		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x) = data_sum;



	}
	else                                                                  //������Ŀ���ڳ�ͷ��Ŀ            
	{
		//data_1 = (tInput + blockIdx.x*iWidth + blockIdx.y - threadIdx.x)->x;
		for (int i = 0; i <= iParaLen - 1; i++)
		{

			data_1 = *(tInput + blockIdx.x*iWidth + threadIdx.x - i);   //b(0)*x(n-0)+b(1)*x(n-1)+...+b(nb-2)*x(n-(nb-2))  

			data_sum += (data_1*param[i]);

		}

		*(tOutPut + blockIdx.x * iWidth + threadIdx.x) = data_sum;

	}




}



   //changed   by   wong      2016/06/22        zero-phase  filter    size  :  299*799

  // ���˲� ��������     


__global__  void   lowpass_back_799(Complex* tInput, int iWidth, float* param, int iParaLen, Complex* tOutPut)   {


	

	float data_1;

	float data_sum;

	data_sum = 0.0;



	if (threadIdx.x <= iParaLen - 1)   {                                //  ���ݳ���С�ڵ����˲�����ͷ����


		for (int i = 0; i <= threadIdx.x; i++)
		{


			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);      //

			data_sum += (data_1*param[i]);

		}


		data_1 = *(tInput + blockIdx.x * iWidth + iWidth - 1);                        //  x(N-1) 


		for (int j = threadIdx.x + 1; j <= iParaLen - 1; j++)
		{

			data_sum += (data_1*param[j]);                                            // b(n+1)*x(N-1)+...+b(nb-1)*x(N-1) 

		}


		*(tOutPut + blockIdx.x * iWidth + iWidth - 1 -threadIdx.x) = data_sum;                 // y(n)

	}

	else    {                                                                       //���ݳ��ȴ����˲�����ͷ����         


		for (int i = 0; i <= iParaLen - 1; i++)
		{
			data_1 = *(tInput + blockIdx.x*iWidth + iWidth - 1 - threadIdx.x + i);

			data_sum += (data_1*param[i]);                                         //  y(N-1-n) = b(0)*x(N-1-n+0) +b(1)*x(N-1-n+1)+...+b(nb-1)*x(N-1-n+nb-1)

		}

		*(tOutPut + blockIdx.x * iWidth + iWidth - 1 - threadIdx.x) = data_sum;




	}



}










	




CvMat*  CudaMain::lowpassFilt_799_cuda(CvMat* disMat)  { 


	Complex* h_MatData = (Complex*)disMat->data.fl;

	hipMemsetAsync(lowBackMat, 0, sizeof(Complex)*disMat->rows*disMat->cols);

	hipMemcpyAsync(lowFrontMat, h_MatData, sizeof(Complex)*disMat->rows*disMat->cols, hipMemcpyHostToDevice);           //����CPU��RF���ݵ�GPU

	int steps = cpu_lowfilterParam.size();

	hipMemcpyAsync(lowfilterParam, &cpu_lowfilterParam[0], sizeof(float)*steps, hipMemcpyHostToDevice);                  //����CPU�г�ͷ���ݵ�GPU 





	dim3 blockID, threadID;

	blockID.x = disMat->rows;

	threadID.x = disMat->cols;


	lowpass_front_799 << <blockID, threadID >> >(lowFrontMat, disMat->cols, lowfilterParam, steps, lowBackMat);

	hipDeviceSynchronize();

	//test   for  lower   begin

	hipMemcpy(h_MatData, lowBackMat, sizeof(Complex)*disMat->cols*disMat->rows, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

	SaveDataFile("799_lower.dat", disMat);

	//test  end 

	hipMemcpy(lowFrontMat, lowBackMat, sizeof(Complex)*disMat->rows*disMat->cols, hipMemcpyDeviceToDevice);


	lowpass_back_799 << <blockID, threadID >> >(lowFrontMat, disMat->cols, lowfilterParam, steps, lowBackMat);


	hipDeviceSynchronize();


	hipFree(lowfilterParam);

	hipMemcpy(h_MatData, lowBackMat, sizeof(Complex)*disMat->cols*disMat->rows, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU

	hipFree(lowFrontMat);

	hipFree(lowBackMat);


	SaveDataFile("lowpass_gpu.dat", disMat);



	const   char* file_pwd = "lowpass_gpu.bmp";


	MakeImage(disMat, file_pwd);

	return disMat;




}


__device__  void    fitLine_cv_func(Complex*xx_tmp, Complex*yy_tmp, Complex* result)   {


	Complex xmean = 0.0f;

	Complex ymean = 0.0f;

	for (int i = 0; i < 5; i++)
	{
		xmean += xx_tmp[i];

		ymean += yy_tmp[i];

	}


	xmean /= 5;

	ymean /= 5;


	Complex sumx2 = 0.0f;

	Complex sumxy = 0.0f;

	for (int i = 0; i < 5; i++)
	{

		sumx2 += (xx_tmp[i] - xmean) * (xx_tmp[i] - xmean);

		sumxy += (yy_tmp[i] - ymean) * (xx_tmp[i] - xmean);

	}


	*result = (Complex)(sumxy / sumx2);

}





//changed   by  wong     2016/07/04
__global__  void  fitLine_L2_cuda(Complex*strain_IN, Complex*xx_IN,  Complex*strainOut)   {

	int   offset = blockIdx.x *blockDim.x + threadIdx.x;                        // ���λ�ƾ���ƫ��ֵ


	int    bid = blockIdx.x;                                                    // block   id

	int    tid = threadIdx.x;                                                   // thread  id     


	int    act_off = blockIdx.x *(blockDim.x + 5 - 1) + threadIdx.x;  // input



	Complex   xx_tmp[5];

	Complex  yy_tmp[5];


	for (int i = 0; i < 5; i++)  {

		xx_tmp[i] = xx_IN[act_off + i];

		yy_tmp[i] = strain_IN[act_off + i];


	}


	fitLine_cv_func(xx_tmp, yy_tmp, &strainOut[offset]);


}






//  changed  by  wong      2016/07/01

void  CudaMain::strainCalculate_cuda(CvMat*disMat,    CvMat* fitMat)  {


	Complex* h_MatData = (Complex*)disMat->data.fl;


	Complex* out_MatData = (Complex*)fitMat->data.fl;


	hipMemcpyAsync(fit_IN, h_MatData, sizeof(Complex)*disMat->rows*disMat->cols, hipMemcpyHostToDevice);           //����CPU��RF���ݵ�GPU


	int   fit_points = 5;


	dim3 blockID, threadID;

	blockID.x = disMat->rows;

	threadID.x = disMat->cols - fit_points +1;


//	Complex* point_num;


//	hipMalloc(&point_num,  sizeof(Complex));             //  λ�ƾ���GPU�ڴ����


	//XXֵ����

	int   xx_rows = disMat->rows;

	int  xx_cols  = disMat->cols;


	CvMat*    xx_mat = cvCreateMat(xx_rows, xx_cols, CV_32FC1);


	   
	for (int i = 0; i < xx_rows; i++)   {

		for (int j = 0; j < xx_cols; j++)  {
		
		
			*(static_cast<float*>(static_cast<void*>(CV_MAT_ELEM_PTR(*xx_mat, i, j)))) = j*fit_points;
				
		}
	
	}


	Complex* xx_IN  ;

    hipMalloc(&xx_IN, sizeof(Complex)*xx_rows*xx_cols);             //  λ�ƾ���GPU�ڴ����


	Complex* xx_Data = (Complex*)xx_mat->data.fl;


	hipMemcpyAsync(xx_IN, xx_Data, sizeof(Complex)*xx_mat->rows*xx_mat->cols, hipMemcpyHostToDevice);



	fitLine_L2_cuda << <blockID, threadID >> >  (fit_IN, xx_IN,  fit_Out);


	hipMemcpy(out_MatData, fit_Out, sizeof(Complex)*fitMat->rows*fitMat->cols, hipMemcpyDeviceToHost);   //����GPU��������ݵ�	CPU




	hipFree(fit_IN);

	hipFree(xx_IN);

	hipFree(fit_Out);


	SaveDataFile("fitLine_gpu.dat", fitMat);







}




void  CudaMain::ImagePostProc(IplImage *strImage, const char *filename, const CvPoint &start, const CvPoint &end)
{

	const char * gray_file = "strain_gpu_gray.bmp";


	{
		IplImage *pimgStrain = cvCreateImage(cvGetSize(strImage), strImage->depth, 3);

		cvCvtColor(strImage, pimgStrain, CV_GRAY2BGR);

		cvSaveImage(gray_file, pimgStrain);

		cvReleaseImage(&pimgStrain);

	}

	{

		IplImage *pImage = cvLoadImage(gray_file, 0);

		IplImage *pimgStrain = cvCreateImage(cvGetSize(pImage), pImage->depth, 3);

		pimgStrain = cvCreateImage(cvGetSize(pImage), pImage->depth, 3);


		//ͼ����ǿ ��1
		// ������� [0,0.5] �� [0.5,1], gamma=1  ͼ����ǿ
		ImageAdjust(pImage, pImage, 0, 0.5, 0, 0.5, 0.6);// Y����mapped to bottom and top of dst	

		//cvSaveImage("res\\ImageAdjust.bmp", image);//������ǿЧ��ͼ

		//ͼ����ǿ ��2 Ч������
		//ImageStretchByHistogram(image, image);//ͼ����ǿ: �����ȫ��������
		//cvSaveImage("res\\ImageStretchByHistogram.bmp", image);

		//ͼ����ǿ ��3 Ч������
		//ImageStretchByHistogram2(image, image);//ͼ����ǿ: �����ȫ��������
		//cvSaveImage("res\\ImageStretchByHistogram2.bmp", image);

		cvNot(pImage, pImage);//�ڰ���ɫ��ת

		//cvSaveImage("res\\cvNot.bmp", image);//�ڰ�ͼ
		cvCvtColor(pImage, pimgStrain, CV_GRAY2BGR);//ͼ��ת����BGR


		ChangeImgColor(pimgStrain);


		cvLine(pimgStrain, start, end, CV_RGB(255, 0, 0), 2, CV_AA, 0);   //����


		cvSaveImage(filename, pimgStrain);


		//�ͷ���Դ
		cvReleaseImage(&pImage);

		cvReleaseImage(&pimgStrain);

	}

}










//////////////////////////////////////////////////////////////////////////
// �����任
// pmatDisplacement,   rows: disp;  cols: time-extent( lines)
//     ��,��ʾһ����, Ҳ����ʱ�� ��
//     ��,��ʾӦ���ֵ
//////////////////////////////////////////////////////////////////////////

void   CudaMain::RadonSum(const CvMat *pmatDisplacement, CvMat **ppmatRodan) {


	int xstart          = 0;

	int xend            = pmatDisplacement->rows;                      //159

	int t               = pmatDisplacement->cols;                     // time extent        //298 

	CvMat *pmatRodan    = cvCreateMat(t - 1, t, pmatDisplacement->type);

	cvZero(pmatRodan);

	int tstart          = 0;

	int tend            = 0;

	int dx              = 0;

	float dt            = 0.0f;

	float c             = 0.0f;


	for (tstart = 0; tstart < t - 1; tstart++)
	{

		for (tend = tstart + 1; tend < t; tend++)
		{

			c = (float)(xend - xstart) / (tend - tstart);                     //k

			for (dx = xstart; dx < xend; dx++)
			{

				dt = tstart + (dx - xstart) / c;                             //

				CV_MAT_ELEM(*pmatRodan, float, tstart, tend) = CV_MAT_ELEM(*pmatRodan, float, tstart, tend)
					+ CV_MAT_ELEM(*pmatDisplacement, float, dx, (int)dt);

			}
		}
	}


	*ppmatRodan = pmatRodan;






}









//�����ֶμ���
void  CudaMain::RadonProcess2(CvPoint &s, CvPoint &e, ConfigParam*config, const CvRect &sub_rc, const CvMat &matStrain)
{

	int  radon_num = config->radon_num;                    // 3
	 

	int  radon_step = config->radon_step;                  // 20



	int  intpl_multiple = 1;                               // ��ֵ��������������任   



	std::vector<RadonParam> array_params;



	for (int i = 0; i < radon_num; i++)                      //3
	{


		RadonParam param;

		param.rc.x = sub_rc.x;

		param.rc.y = sub_rc.y + i*radon_step;

		param.rc.width = sub_rc.width;

		param.rc.height = sub_rc.height;


		CvMat *pmatSub = cvCreateMatHeader(param.rc.height-1, param.rc.width-1, matStrain.type);


		cvGetSubRect(&matStrain, pmatSub, cvRect(param.rc.x, param.rc.y, param.rc.width-1, param.rc.height-1));


		CvMat *pmatRadon = 0;


		CvMat *pmatMultiple = cvCreateMat(pmatSub->rows, pmatSub->cols * intpl_multiple, pmatSub->type);


		cvResize(pmatSub, pmatMultiple);


		RadonSum(pmatMultiple, &pmatRadon);


		double  min_val;


		double  max_val;


		CvPoint min_loc;


		CvPoint max_loc;


		cvMinMaxLoc(pmatRadon, &min_val, &max_val, &min_loc, &max_loc);


		param.pt = max_loc;


		param.xWidth = param.pt.y - param.pt.x;//add by wxm


		array_params.push_back(param);


		cvReleaseMat(&pmatRadon);


		cvReleaseMat(&pmatMultiple);


		cvReleaseMatHeader(&pmatSub);


	}


	std::sort(array_params.begin(), array_params.end(), MyLessThan2());



	if (config->calc_type.compare("middle") == 0)
	{

		int size = array_params.size();


		s.x = array_params[size / 2].pt.y / intpl_multiple;


		s.y = array_params[size / 2].rc.y;


		e.x = array_params[size / 2].pt.x / intpl_multiple;


		e.y = array_params[size / 2].rc.y + array_params[size / 2].rc.height-1;

	}

	else if (config->calc_type.compare("max") == 0)
	{

		int size = array_params.size();

		s.x = array_params[0].pt.y / intpl_multiple;

		s.y = array_params[0].rc.y;


		e.x = array_params[0].pt.x / intpl_multiple;

		e.y = array_params[0].rc.y + array_params[0].rc.height-1;

	}

	else if (config->calc_type.compare("min") == 0)

	{

		int size = array_params.size();

		s.x = array_params[size - 1].pt.y / intpl_multiple;

		s.y = array_params[size - 1].rc.y;


		e.x = array_params[size - 1].pt.x / intpl_multiple;

		e.y = array_params[size - 1].rc.y + array_params[size - 1].rc.height-1;

	}

	else
	{
		//

	}










}






//�����任&����в�&����ģ��

void    CudaMain::random_proess_cuda(CvMat*fitMat, ConfigParam*config, EOutput &output)  {

	
	

		int    win_size          = config->windowHW;                                              //  ���ڴ�С


		double overlap           = (config->windowHW - config->step) / (float)config->windowHW;  //   �غ��ʣ�90%

		double sound_velocity    = config->acousVel;                                             //   �����ٶ�


		double sample_frq        = config->sampleFreqs;                                         //    ������                      

		double prf               = 1 / 300e-6;                                                  //    �ظ���


		int    dep_start         = (config->sb_x < 0) ? 0 : config->sb_x;

		int    dep_size          = (config->sb_w < 0) ? fitMat->width : config->sb_w;

		int    dep_end           = dep_start + dep_size - 1;

		int    t_start           = (config->sb_y < 0) ? 0 : config->sb_y;

		int    t_size            = (config->sb_h < 0) ? fitMat->rows : config->sb_h;

		int    t_end             = t_start + t_size - 1;


		CvMat *pmatStrainTran    = cvCreateMat(fitMat->cols, fitMat->rows, fitMat->type);       // ��strainMatת��      795*299


		cvTranspose(fitMat, pmatStrainTran);


		CvPoint                   start;

		CvPoint                    end;
		
		CvRect                     rect;


		rect.x                    = t_start;

		rect.y                    = dep_start;

		rect.width                = t_size;

		rect.height               = dep_size;


//		rect.left                = t_start;

//		rect.right               = t_end;

//		rect.top                 = dep_start;

//		rect.bottom              = dep_end;

		
#if 1
		RadonProcess2(start, end, config ,rect, *pmatStrainTran);
#endif






		double v                  = ((end.y - start.y) * win_size * (1 - overlap) * sound_velocity / sample_frq / 2)
			/ ((end.x - start.x) / prf);



		double e                  = v * v * 3;


		output.v                  = (float)v;


		output.e                  = (float)e;

		cvReleaseMat(&pmatStrainTran);


		
    // ����б��    �е�����   changed  by  wong    2016/07/08

		/*
		IplImage *strImage = cvCreateImage(cvSize(fitMat->cols, fitMat->rows), IPL_DEPTH_32F, 1);     //������ʾӦ��, �����outDataMat����ת��,���еߵ�.


		for (int i = 0; i < strImage->width; i++) {

			for (int j = 0; j < strImage->height; j++)	{


				float*	tmp = static_cast<float*>(static_cast<void*>(strImage->imageData + j * strImage->widthStep + sizeof(float) * i));  //ȡӦ��ͼ���Ӧλ��


				*tmp = 100 * CV_MAT_ELEM(*fitMat, float, i, j);

			}

		}


		    char* filename = "strain_gpu.bmp";


			ImagePostProc(strImage, filename, start, end);


			cvReleaseImage(&strImage);

   */


}












void  CudaMain::process(const EInput &input, EOutput& output) {

//	mallocMem();                                                                                 // �����ڴ�

//	inputRfData(input);                                                                          // ��ȡRF����,��cpu_inputMat 

//	inputConfigParam(config);                                                                    // ���ò��� ��cpu_config

// getFilterParam(config->bpfilt_file);                                                          // ��ȡ�˲�����ͨ��������cpu_filterParam




	   bandpassFilt_1024_cuda(cpu_inputMat);                                                     // ��ͨ�˲�       


	   int  multiWin    = 2;

	   int winSize      = cpu_config->windowHW;

	   int  stepSize    = cpu_config->step;


	   
	    computeDisplacement_cuda(cpu_inputMat, multiWin, winSize, stepSize, cpu_disMat);       //  λ�Ƽ���       
	

		lowpassFilt_799_cuda (cpu_disMat);                                                    //    ��ͨ�˲�  


	



		

		strainCalculate_cuda(cpu_disMat,   cpu_fitMat);                                       //  ֱ�����



		


		random_proess_cuda(cpu_fitMat, cpu_config, output);                                 //  �����任�����ٶȺ�����ģ��




		int   ss = 0;










	


}



